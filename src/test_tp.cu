#include "hip/hip_runtime.h"
/**
 **
 ** dtt8x8.cu - CPU test code to run GPU tile processor
 **
 ** Copyright (C) 2018 Elphel, Inc.
 **
 ** -----------------------------------------------------------------------------**
 **
 **  dtt8x8.cu is free software: you can redistribute it and/or modify
 **  it under the terms of the GNU General Public License as published by
 **  the Free Software Foundation, either version 3 of the License, or
 **  (at your option) any later version.
 **
 **  This program is distributed in the hope that it will be useful,
 **  but WITHOUT ANY WARRANTY; without even the implied warranty of
 **  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 **  GNU General Public License for more details.
 **
 **  You should have received a copy of the GNU General Public License
 **  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 **
 **  Additional permission under GNU GPL version 3 section 7
 **
 **  If you modify this Program, or any covered work, by linking or
 **  combining it with NVIDIA Corporation's CUDA libraries from the
 **  NVIDIA CUDA Toolkit (or a modified version of those libraries),
 **  containing parts covered by the terms of NVIDIA CUDA Toolkit
 **  EULA, the licensors of this Program grant you additional
 **  permission to convey the resulting work.
 ** -----------------------------------------------------------------------------**
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// for reading binary files
#include <fstream>
#include <iterator>
#include <vector>

//#include "dtt8x8.cuh"
#include "dtt8x8.h"
#include "TileProcessor.cuh"
///#include "hip/hip_runtime_api.h"
//#include "cudaProfiler.h"


float * copyalloc_kernel_gpu(float * kernel_host,
		                int size, // size in floats
						int full_size)
{
	float *kernel_gpu;
    checkCudaErrors(hipMalloc((void **)&kernel_gpu, full_size * sizeof(float)));
    checkCudaErrors(hipMemcpy( // segfault
    		kernel_gpu,
    		kernel_host,
			size * sizeof(float),
            hipMemcpyHostToDevice));
    return kernel_gpu;
}

float * copyalloc_kernel_gpu(float * kernel_host,
		                int size)
{
	return copyalloc_kernel_gpu(kernel_host,
            size, // size in floats
			size);
}



float * alloccopy_from_gpu(
		float * gpu_data,
		float * cpu_data, // if null, will allocate
		int size)
{
	if (!cpu_data) {
		cpu_data = (float *)malloc(size*sizeof(float));
	}
	checkCudaErrors(hipMemcpy( // segfault
			cpu_data,
			gpu_data,
			size * sizeof(float),
			hipMemcpyDeviceToHost));

	return cpu_data;
}


float * alloc_kernel_gpu(int size) // size in floats
{
	float *kernel_gpu;
    checkCudaErrors(hipMalloc((void **)&kernel_gpu, size * sizeof(float)));
    return kernel_gpu;
}


float ** copyalloc_pointers_gpu(float ** gpu_pointer,
		                int size) // number of entries (cameras)
{
	float ** gpu_pointer_to_gpu_pointers;
    checkCudaErrors(hipMalloc((void **)&gpu_pointer_to_gpu_pointers, size * sizeof(float*)));
    checkCudaErrors(hipMemcpy(
    		gpu_pointer_to_gpu_pointers,
			gpu_pointer,
			size * sizeof(float*),
            hipMemcpyHostToDevice));
    return gpu_pointer_to_gpu_pointers;
}


float * copyalloc_image_gpu(float * image_host,
						size_t* dstride, // in floats !
		                int width,
						int height)
{
	float *image_gpu;
    checkCudaErrors(hipMallocPitch((void **)&image_gpu, dstride, width * sizeof(float), height));
    checkCudaErrors(hipMemcpy2D(
    		image_gpu,
            *dstride, //  * sizeof(float),
			image_host,
			width * sizeof(float), // make in 16*n?
            width * sizeof(float),
			height,
			hipMemcpyHostToDevice));
    return image_gpu;
}

float * alloc_image_gpu(size_t* dstride, // in bytes!!
		                int width,
						int height)
{
	float *image_gpu;
    checkCudaErrors(hipMallocPitch((void **)&image_gpu, dstride, width * sizeof(float), height));
    return image_gpu;
}

int readFloatsFromFile(float *       data, // allocated array
					   const char *  path) // file path
{

    std::ifstream input(path, std::ios::binary );
    // copies all data into buffer
    std::vector<char> buffer((
            std::istreambuf_iterator<char>(input)),
            (std::istreambuf_iterator<char>()));
    std::copy( buffer.begin(), buffer.end(), (char *) data);
	return 0;
}
int writeFloatsToFile(float *       data, // allocated array
		               int           size, // length in elements
					   const char *  path) // file path
{

//  std::ifstream input(path, std::ios::binary );
	std::ofstream ofile(path, std::ios::binary);
	ofile.write((char *) data, size * sizeof(float));
	return 0;
}

// Prepare low pass filter (64 long) to be applied to each quadrant of the CLT data
void set_clt_lpf(
		float * lpf,    // size*size array to be filled out
		float   sigma,
		const int     dct_size)
{
	int dct_len = dct_size * dct_size;
	if (sigma == 0.0f) {
		lpf[0] = 1.0f;
		for (int i = 1; i < dct_len; i++){
			lpf[i] = 0.0;
		}
	} else {
		for (int i = 0; i < dct_size; i++){
			for (int j = 0; j < dct_size; j++){
				lpf[i*dct_size+j] = exp(-(i*i+j*j)/(2*sigma));
			}
		}
		// normalize
		double sum = 0;
		for (int i = 0; i < dct_size; i++){
			for (int j = 0; j < dct_size; j++){
				double d = 	lpf[i*dct_size+j];
				d*=cos(M_PI*i/(2*dct_size))*cos(M_PI*j/(2*dct_size));
				if (i > 0) d*= 2.0;
				if (j > 0) d*= 2.0;
				sum +=d;
			}
		}
		for (int i = 0; i< dct_len; i++){
			lpf[i] /= sum;
		}
	}
}



/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/


int main(int argc, char **argv)
{
    //
    // Sample initialization
    //
    printf("%s Starting...\n\n", argv[0]);
    printf("sizeof(float*)=%d\n",(int)sizeof(float*));

    //initialize CUDA
    findCudaDevice(argc, (const char **)argv);

    // CLT testing

    const char* kernel_file[] = {
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0_transposed.kernel",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1_transposed.kernel",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2_transposed.kernel",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3_transposed.kernel"};

    const char* kernel_offs_file[] = {
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0_transposed.kernel_offsets",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1_transposed.kernel_offsets",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2_transposed.kernel_offsets",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3_transposed.kernel_offsets"};

    const char* image_files[] = {
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0.bayer",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1.bayer",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2.bayer",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3.bayer"};

    const char* ports_offs_xy_file[] = {
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0.portsxy",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1.portsxy",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2.portsxy",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3.portsxy"};

    const char* ports_clt_file[] = { // never referenced
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0.clt",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1.clt",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2.clt",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3.clt"};
    const char* result_rbg_file[] = {
    		"/data_ssd/git/tile_processor_gpu/clt/main_chn0.rbg",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn1.rbg",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn2.rbg",
			"/data_ssd/git/tile_processor_gpu/clt/main_chn3.rbg"};
    const char* result_corr_file = "/data_ssd/git/tile_processor_gpu/clt/main_corr.corr";
    const char* result_textures_file =       "/data_ssd/git/tile_processor_gpu/clt/texture.rgba";
    const char* result_textures_rgba_file = "/data_ssd/git/tile_processor_gpu/clt/texture_rgba.rgba";
    // not yet used
    float lpf_sigmas[3] = {0.9f, 0.9f, 0.9f}; // G, B, G

    float port_offsets[NUM_CAMS][2] =  {// used only in textures to scale differences
			{-0.5, -0.5},
			{ 0.5, -0.5},
			{-0.5,  0.5},
			{ 0.5,  0.5}};

    int keep_texture_weights = 1; // try with 0 also
    int texture_colors = 3; // result will be 3+1 RGBA (for mono - 2)


/*
#define IMG_WIDTH    2592
#define IMG_HEIGHT   1936
#define NUM_CAMS        4
#define NUM_COLORS      3
#define KERNELS_STEP   16
#define KERNELS_HOR   164
#define KERNELS_VERT  123
#define KERNEL_OFFSETS  8
#define TILESX        324
#define TILESY        242
*/
/*
    struct tp_task {
    	long task;
		short ty;
		short tx;
		float xy[NUM_CAMS][2];
    } ;
*/
    int KERN_TILES = KERNELS_HOR *  KERNELS_VERT * NUM_COLORS;
    int KERN_SIZE =  KERN_TILES * 4 * 64;

//    int CORR_SIZE = (2 * DTT_SIZE -1) * (2 * DTT_SIZE -1);
    int CORR_SIZE = (2 * CORR_OUT_RAD + 1) * (2 * CORR_OUT_RAD + 1);



    float            * host_kern_buf =  (float *)malloc(KERN_SIZE * sizeof(float));

    struct tp_task     task_data [TILESX*TILESY]; // maximal length - each tile
    int                corr_indices         [NUM_PAIRS*TILESX*TILESY];
//    int                texture_indices      [TILESX*TILESY];
    int                texture_indices      [TILESX*TILESYA];
    int                cpu_woi              [4];

    // host array of pointers to GPU memory
    float            * gpu_kernels_h        [NUM_CAMS];
    struct CltExtra  * gpu_kernel_offsets_h [NUM_CAMS];
    float            * gpu_images_h         [NUM_CAMS];
    float              tile_coords_h        [NUM_CAMS][TILESX * TILESY][2];
    float            * gpu_clt_h            [NUM_CAMS];
    float            * gpu_lpf_h            [NUM_COLORS]; // never used
#ifndef NOICLT
    float            * gpu_corr_images_h    [NUM_CAMS];
#endif

    float            * gpu_corrs;
    int              * gpu_corr_indices;

    float            * gpu_textures;
    float            * gpu_textures_rbga;
    int              * gpu_texture_indices;
    int              * gpu_woi;
    int              * gpu_num_texture_tiles;
    float            * gpu_port_offsets;
    int                num_corrs;
    int                num_textures;
    int                num_ports = NUM_CAMS;
    // GPU pointers to GPU pointers to memory
    float           ** gpu_kernels; //           [NUM_CAMS];
    struct CltExtra ** gpu_kernel_offsets; //    [NUM_CAMS];
    float           ** gpu_images; //            [NUM_CAMS];
    float           ** gpu_clt;    //           [NUM_CAMS];
    float           ** gpu_lpf;    //           [NUM_CAMS]; // never referenced

    // GPU pointers to GPU memory
//    float * gpu_tasks;
    struct tp_task  * gpu_tasks;
    size_t  dstride;          // in bytes !
    size_t  dstride_rslt;     // in bytes !
    size_t  dstride_corr;     // in bytes ! for one 2d phase correlation (padded 15x15x4 bytes)
    size_t  dstride_textures; // in bytes ! for one rgba/ya 16x16 tile
    size_t  dstride_textures_rbga; // in bytes ! for one rgba/ya 16x16 tile


    float lpf_rbg[3][64]; // not used
    for (int ncol = 0; ncol < 3; ncol++) {
    	if (lpf_sigmas[ncol] > 0.0) {
    		set_clt_lpf (
    				lpf_rbg[ncol], // float * lpf,    // size*size array to be filled out
					lpf_sigmas[ncol], // float   sigma,
					8); // int     dct_size)
    		gpu_lpf_h[ncol] = copyalloc_kernel_gpu(lpf_rbg[ncol], 64);
    	} else {
    		gpu_lpf_h[ncol] = NULL;
    	}
    }

    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
        readFloatsFromFile(
        		host_kern_buf, // float * data, // allocated array
				kernel_file[ncam]); // 			   char *  path) // file path
        gpu_kernels_h[ncam] = copyalloc_kernel_gpu(host_kern_buf, KERN_SIZE);

        readFloatsFromFile(
        		host_kern_buf, // float * data, // allocated array
				kernel_offs_file[ncam]); // 			   char *  path) // file path
        gpu_kernel_offsets_h[ncam] = (struct CltExtra *) copyalloc_kernel_gpu(
        		host_kern_buf,
				KERN_TILES * (sizeof( struct CltExtra)/sizeof(float)));
        // will get results back
        gpu_clt_h[ncam] = alloc_kernel_gpu(TILESY * TILESX * NUM_COLORS * 4 * DTT_SIZE * DTT_SIZE);
        printf("Allocating GPU memory, 0x%x floats\n", (TILESY * TILESX * NUM_COLORS * 4 * DTT_SIZE * DTT_SIZE)) ;
        // allocate result images (3x height to accommodate 3 colors

        // Image is extended by 4 pixels each side to avoid checking (mclt tiles extend by 4)
        //host array of pointers to GPU arrays
#ifndef NOICLT
        gpu_corr_images_h[ncam] = alloc_image_gpu(
        		&dstride_rslt,                // size_t* dstride, // in bytes!!
				IMG_WIDTH + DTT_SIZE,         // int width,
				3*(IMG_HEIGHT + DTT_SIZE));   // int height);
#endif
    }
    // allocates one correlation kernel per line (15x15 floats), number of rows - number of tiles * number of pairs
    gpu_corrs = alloc_image_gpu(
    		&dstride_corr,                  // in bytes ! for one 2d phase correlation (padded 15x15x4 bytes)
			CORR_SIZE,                      // int width,
			NUM_PAIRS * TILESX * TILESY);   // int height);
    // read channel images (assuming host_kern_buf size > image size, reusing it)
    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
        readFloatsFromFile(
        		host_kern_buf, // float * data, // allocated array
				image_files[ncam]); // 			   char *  path) // file path
        gpu_images_h[ncam] =  copyalloc_image_gpu(
        		host_kern_buf, // float * image_host,
				&dstride,      // size_t* dstride,
				IMG_WIDTH,     // int width,
				IMG_HEIGHT);   // int height);
    }
//#define DBG_TILE  (174*324 +118)

    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
        readFloatsFromFile(
			    (float *) &tile_coords_h[ncam],
				ports_offs_xy_file[ncam]); // 			   char *  path) // file path
    }

    // build TP task that processes all tiles in linescan order
    for (int ty = 0; ty < TILESY; ty++){
        for (int tx = 0; tx < TILESX; tx++){
            int nt = ty * TILESX + tx;
            task_data[nt].task = 0xf | (((1 << NUM_PAIRS)-1) << TASK_CORR_BITS);
            task_data[nt].txy = tx + (ty << 16);
            for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
                task_data[nt].xy[ncam][0] = tile_coords_h[ncam][nt][0];
                task_data[nt].xy[ncam][1] = tile_coords_h[ncam][nt][1];
            }
        }
    }

    int tp_task_size =  sizeof(task_data)/sizeof(struct tp_task);


#ifdef DBG0
//#define NUM_TEST_TILES 128
#define NUM_TEST_TILES 1
    for (int t = 0; t < NUM_TEST_TILES; t++) {
    	task_data[t].task = 1;
    	task_data[t].txy = ((DBG_TILE + t) - 324* ((DBG_TILE + t) / 324)) + (((DBG_TILE + t) / 324)) << 16;
    	int nt = task_data[t].ty * TILESX + task_data[t].tx;

    	for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    		task_data[t].xy[ncam][0] = tile_coords_h[ncam][nt][0];
    		task_data[t].xy[ncam][1] = tile_coords_h[ncam][nt][1];
    	}
    }
    tp_task_size =  NUM_TEST_TILES; // sizeof(task_data)/sizeof(float);

#endif

    // segfault in the next
    gpu_tasks = (struct tp_task  *) copyalloc_kernel_gpu((float * ) &task_data, tp_task_size * (sizeof(struct tp_task)/sizeof(float)));

    // build corr_indices
    num_corrs = 0;
    for (int ty = 0; ty < TILESY; ty++){
    	for (int tx = 0; tx < TILESX; tx++){
    		int nt = ty * TILESX + tx;
    		int cm = (task_data[nt].task >> TASK_CORR_BITS) & ((1 << NUM_PAIRS)-1);
    		if (cm){
    			for (int b = 0; b < NUM_PAIRS; b++) if ((cm & (1 << b)) != 0) {
    				corr_indices[num_corrs++] = (nt << CORR_NTILE_SHIFT) | b;
    			}
    		}
    	}
    }
    // num_corrs now has the total number of correlations
    // copy corr_indices to gpu
//    gpu_corr_indices = (int  *) copyalloc_kernel_gpu((float * ) corr_indices, num_corrs);
    gpu_corr_indices = (int  *) copyalloc_kernel_gpu(
    		(float * ) corr_indices,
			num_corrs,
			NUM_PAIRS * TILESX * TILESY);

    // build texture_indices
    num_textures = 0;
    for (int ty = 0; ty < TILESY; ty++){
    	for (int tx = 0; tx < TILESX; tx++){
    		int nt = ty * TILESX + tx;
//    		int cm = (task_data[nt].task >> TASK_TEXTURE_BIT) & 1;
    		int cm = task_data[nt].task & TASK_TEXTURE_BITS;
    		if (cm){
    			texture_indices[num_textures++] = (nt << CORR_NTILE_SHIFT) | (1 << LIST_TEXTURE_BIT);
    		}
    	}
    }
    // num_textures now has the total number of textures
    // copy corr_indices to gpu
//  gpu_texture_indices = (int  *) copyalloc_kernel_gpu((float * ) texture_indices, num_textures);
    gpu_texture_indices = (int  *) copyalloc_kernel_gpu(
    		(float * ) texture_indices,
			num_textures,
			TILESX * TILESYA); // number of rows - multiple of 4
    // just allocate
    checkCudaErrors(hipMalloc((void **)&gpu_woi,               4 * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&gpu_num_texture_tiles, 8 * sizeof(float))); // for each subsequence - number of non-border,
    // number of border tiles

    // copy port indices to gpu
    gpu_port_offsets = (float *) copyalloc_kernel_gpu((float * ) port_offsets, num_ports * 2);



//    int keep_texture_weights = 1; // try with 0 also
//    int texture_colors = 3; // result will be 3+1 RGBA (for mono - 2)

//		double [][] rgba = new double[numcol + 1 + (keep_weights?(ports + numcol + 1):0)][];

    int tile_texture_size = (texture_colors + 1 + (keep_texture_weights? (NUM_CAMS + texture_colors + 1): 0)) *256;

    gpu_textures = alloc_image_gpu(
    		&dstride_textures,              // in bytes ! for one rgba/ya 16x16 tile
			tile_texture_size,              // int width (floats),
			TILESX * TILESY);               // int height);

    int rgba_width =   (TILESX+1) * DTT_SIZE;
    int rgba_height =  (TILESY+1) * DTT_SIZE;
    int rbga_slices =  texture_colors + 1; // 4/1

    gpu_textures_rbga = alloc_image_gpu(
    		&dstride_textures_rbga,              // in bytes ! for one rgba/ya 16x16 tile
			rgba_width,              // int width (floats),
			rgba_height * rbga_slices);               // int height);


    // Now copy arrays of per-camera pointers to GPU memory to GPU itself

    gpu_kernels =        copyalloc_pointers_gpu (gpu_kernels_h,     NUM_CAMS);
    gpu_kernel_offsets = (struct CltExtra **) copyalloc_pointers_gpu ((float **) gpu_kernel_offsets_h, NUM_CAMS);
    gpu_images =         copyalloc_pointers_gpu (gpu_images_h,      NUM_CAMS);
    gpu_clt =            copyalloc_pointers_gpu (gpu_clt_h,         NUM_CAMS);
//    gpu_corr_images =    copyalloc_pointers_gpu (gpu_corr_images_h, NUM_CAMS);

    //create and start CUDA timer
    StopWatchInterface *timerTP = 0;
    sdkCreateTimer(&timerTP);


    dim3 threads_tp(THREADSX, TILES_PER_BLOCK, 1);
    dim3 grid_tp((tp_task_size + TILES_PER_BLOCK -1 )/TILES_PER_BLOCK, 1);
    printf("threads_tp=(%d, %d, %d)\n",threads_tp.x,threads_tp.y,threads_tp.z);
    printf("grid_tp=   (%d, %d, %d)\n",grid_tp.x,   grid_tp.y,   grid_tp.z);

#ifdef DBG_TILE
    const int numIterations = 1; //0;
    const int i0 =  0; // -1;
#else
    const int numIterations = 10; // 0; //0;
    const int i0 = -1; // 0; // -1;
#endif
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(convert_correct_tiles), hipFuncCachePreferShared);
///    hipProfilerStart();
    float ** fgpu_kernel_offsets = (float **) gpu_kernel_offsets; //    [NUM_CAMS];

    for (int i = i0; i < numIterations; i++)
    {
        if (i == 0)
        {
            checkCudaErrors(hipDeviceSynchronize());
            sdkResetTimer(&timerTP);
            sdkStartTimer(&timerTP);
        }

        convert_correct_tiles<<<grid_tp,threads_tp>>>(
        		fgpu_kernel_offsets,    // struct CltExtra      ** gpu_kernel_offsets,
				gpu_kernels,           // 		float           ** gpu_kernels,
				gpu_images,            // 		float           ** gpu_images,
				gpu_tasks,             // 		struct tp_task  * gpu_tasks,
				gpu_clt,               //       float           ** gpu_clt,            // [NUM_CAMS][TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
				dstride/sizeof(float), // 		size_t            dstride, // for gpu_images
				tp_task_size,          // 		int               num_tiles) // number of tiles in task
				0); // 7); // 0); // 7);                    //       int               lpf_mask)            // apply lpf to colors : bit 0 - red, bit 1 - blue, bit2 - green


        getLastCudaError("Kernel execution failed");
        checkCudaErrors(hipDeviceSynchronize());
        printf("%d\n",i);
    }
//    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timerTP);
    float avgTime = (float)sdkGetTimerValue(&timerTP) / (float)numIterations;
    sdkDeleteTimer(&timerTP);
    printf("Run time =%f ms\n",  avgTime);


#ifdef SAVE_CLT
    int rslt_size = (TILESY * TILESX * NUM_COLORS * 4 * DTT_SIZE * DTT_SIZE);
    float * cpu_clt = (float *)malloc(rslt_size*sizeof(float));
    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    	checkCudaErrors(hipMemcpy( // segfault
    			cpu_clt,
				gpu_clt_h[ncam],
				rslt_size * sizeof(float),
    			hipMemcpyDeviceToHost));
#ifndef DBG_TILE
        printf("Writing CLT data to %s\n",  ports_clt_file[ncam]);
    	writeFloatsToFile(cpu_clt, // float *       data, // allocated array
    			rslt_size, // int           size, // length in elements
				ports_clt_file[ncam]); // 			   const char *  path) // file path
#endif
    }
#endif

#ifdef TEST_IMCLT
     {
    	// testing imclt
    	dim3 threads_imclt(IMCLT_THREADS_PER_TILE, IMCLT_TILES_PER_BLOCK, 1);
    	dim3 grid_imclt(1,1,1);
    	printf("threads_imclt=(%d, %d, %d)\n",threads_imclt.x,threads_imclt.y,threads_imclt.z);
    	printf("grid_imclt=   (%d, %d, %d)\n",grid_imclt.x,   grid_imclt.y,   grid_imclt.z);
    	for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    		test_imclt<<<grid_imclt,threads_imclt>>>(
    				gpu_clt_h[ncam], // ncam]); //                //       float           ** gpu_clt,            // [NUM_CAMS][TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
					ncam);                                        // int             ncam); // just for debug print
    	}
    	getLastCudaError("Kernel execution failed");
    	checkCudaErrors(hipDeviceSynchronize());
    	printf("test_imclt() DONE\n");
    }
#endif


#ifndef NOICLT
    // testing imclt
    dim3 threads_imclt(IMCLT_THREADS_PER_TILE, IMCLT_TILES_PER_BLOCK, 1);
    printf("threads_imclt=(%d, %d, %d)\n",threads_imclt.x,threads_imclt.y,threads_imclt.z);
    StopWatchInterface *timerIMCLT = 0;
    sdkCreateTimer(&timerIMCLT);

    for (int i = i0; i < numIterations; i++)
    {
    	if (i == 0)
    	{
    		checkCudaErrors(hipDeviceSynchronize());
    		sdkResetTimer(&timerIMCLT);
    		sdkStartTimer(&timerIMCLT);
    	}

    	for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    		for (int color = 0; color < NUM_COLORS; color++) {
#ifdef IMCLT14
    			for (int v_offs = 0; v_offs < 1; v_offs++){     // temporarily for debugging
    				for (int h_offs = 0; h_offs < 1; h_offs++){ // temporarily for debugging
#else
    	    			for (int v_offs = 0; v_offs < 2; v_offs++){
    	    				for (int h_offs = 0; h_offs < 2; h_offs++){
#endif
    					int tilesy_half = (TILESY + (v_offs ^ 1)) >> 1;
    					int tilesx_half = (TILESX + (h_offs ^ 1)) >> 1;
    					int tiles_in_pass = tilesy_half * tilesx_half;
    					dim3 grid_imclt((tiles_in_pass + IMCLT_TILES_PER_BLOCK-1) / IMCLT_TILES_PER_BLOCK,1,1);
    					//    				printf("grid_imclt=   (%d, %d, %d)\n",grid_imclt.x,   grid_imclt.y,   grid_imclt.z);
    					imclt_rbg<<<grid_imclt,threads_imclt>>>(
    							gpu_clt_h[ncam], // float           * gpu_clt,            // [TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
								gpu_corr_images_h[ncam], // float           * gpu_rbg,            // WIDTH, 3 * HEIGHT
								color, // int               color,
								v_offs, // int               v_offset,
								h_offs, // int               h_offset,
								dstride_rslt/sizeof(float));            //const size_t      dstride);            // in floats (pixels)
    				}
    			}
    		}
    	}
    	getLastCudaError("Kernel failure");
    	checkCudaErrors(hipDeviceSynchronize());
    	printf("test pass: %d\n",i);
    }

    sdkStopTimer(&timerIMCLT);
    float avgTimeIMCLT = (float)sdkGetTimerValue(&timerIMCLT) / (float)numIterations;
    sdkDeleteTimer(&timerIMCLT);
    printf("Average IMCLT run time =%f ms\n",  avgTimeIMCLT);

    int rslt_img_size =       NUM_COLORS * (IMG_HEIGHT + DTT_SIZE) * (IMG_WIDTH + DTT_SIZE);
    float * cpu_corr_image = (float *)malloc(rslt_img_size * sizeof(float));



    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    	checkCudaErrors(hipMemcpy2D( // segfault
    			cpu_corr_image,
				(IMG_WIDTH + DTT_SIZE) * sizeof(float),
				gpu_corr_images_h[ncam],
				dstride_rslt,
				(IMG_WIDTH + DTT_SIZE) * sizeof(float),
				3* (IMG_HEIGHT + DTT_SIZE),
    			hipMemcpyDeviceToHost));

#ifndef DBG_TILE
        printf("Writing RBG data to %s\n",  result_rbg_file[ncam]);
    	writeFloatsToFile( // will have margins
    			cpu_corr_image, // float *       data, // allocated array
				rslt_img_size, // int           size, // length in elements
				result_rbg_file[ncam]); // 			   const char *  path) // file path
#endif
    }

    free(cpu_corr_image);
#endif


#ifndef NOCORR
//    hipProfilerStart();
    // testing corr
    dim3 threads_corr(CORR_THREADS_PER_TILE, CORR_TILES_PER_BLOCK, 1);
    printf("threads_corr=(%d, %d, %d)\n",threads_corr.x,threads_corr.y,threads_corr.z);
    StopWatchInterface *timerCORR = 0;
    sdkCreateTimer(&timerCORR);

    for (int i = i0; i < numIterations; i++)
    {
    	if (i == 0)
    	{
    		checkCudaErrors(hipDeviceSynchronize());
    		sdkResetTimer(&timerCORR);
    		sdkStartTimer(&timerCORR);
    	}

        dim3 grid_corr((num_corrs + CORR_TILES_PER_BLOCK-1) / CORR_TILES_PER_BLOCK,1,1);
        correlate2D<<<grid_corr,threads_corr>>>(
		gpu_clt,   // float          ** gpu_clt,            // [NUM_CAMS] ->[TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
		3,         // int               colors,             // number of colors (3/1)
		0.25,      // float             scale0,             // scale for R
		0.25,      // float             scale1,             // scale for B
		0.5,       // float             scale2,             // scale for G
		30.0,      // float             fat_zero,           // here - absolute
		num_corrs, // size_t            num_corr_tiles,     // number of correlation tiles to process
		gpu_corr_indices, //  int             * gpu_corr_indices,   // packed tile+pair
		dstride_corr/sizeof(float), // const size_t      corr_stride,        // in floats
		CORR_OUT_RAD, // int               corr_radius,        // radius of the output correlation (7 for 15x15)
		gpu_corrs); // float           * gpu_corrs);          // correlation output data
    	getLastCudaError("Kernel failure");
    	checkCudaErrors(hipDeviceSynchronize());
    	printf("test pass: %d\n",i);
    }

    sdkStopTimer(&timerCORR);
    float avgTimeCORR = (float)sdkGetTimerValue(&timerCORR) / (float)numIterations;
    sdkDeleteTimer(&timerCORR);
    printf("Average CORR run time =%f ms\n",  avgTimeCORR);

    int corr_size =        2 * CORR_OUT_RAD + 1;
    int rslt_corr_size =   num_corrs * corr_size * corr_size;
    float * cpu_corr = (float *)malloc(rslt_corr_size * sizeof(float));



    checkCudaErrors(hipMemcpy2D(
    		cpu_corr,
			(corr_size * corr_size) * sizeof(float),
			gpu_corrs,
			dstride_corr,
			(corr_size * corr_size) * sizeof(float),
			num_corrs,
    		hipMemcpyDeviceToHost));

#ifndef NSAVE_CORR
    		printf("Writing phase correlation data to %s\n",  result_corr_file);
    		writeFloatsToFile(
    				cpu_corr,    // float *       data, // allocated array
					rslt_corr_size,    // int           size, // length in elements
					result_corr_file); // 			   const char *  path) // file path
#endif
    		free(cpu_corr);
#endif // ifndef NOCORR


// -----------------

#ifndef NOTEXTURES
//    hipProfilerStart();
    // testing textures
    dim3 threads_texture(TEXTURE_THREADS_PER_TILE, NUM_CAMS, 1); // TEXTURE_TILES_PER_BLOCK, 1);
    dim3 grid_texture((num_textures + TEXTURE_TILES_PER_BLOCK-1) / TEXTURE_TILES_PER_BLOCK,1,1);
    printf("threads_texture=(%d, %d, %d)\n",threads_texture.x,threads_texture.y,threads_texture.z);
    printf("grid_texture=(%d, %d, %d)\n",grid_texture.x,grid_texture.y,grid_texture.z);
    StopWatchInterface *timerTEXTURE = 0;
    sdkCreateTimer(&timerTEXTURE);

    for (int i = i0; i < numIterations; i++)
    {
    	if (i == 0)
    	{
    		checkCudaErrors(hipDeviceSynchronize());
    		sdkResetTimer(&timerTEXTURE);
    		sdkStartTimer(&timerTEXTURE);
    	}

		// Channel0 weight = 0.294118
		// Channel1 weight = 0.117647
		// Channel2 weight = 0.588235
    	textures_accumulate<<<grid_texture,threads_texture>>> (
//    			0,          // int               border_tile,        // if 1 - watch for border
    			(int *) 0,  //      int             * woi,                // x, y, width,height
		        gpu_clt ,              // float          ** gpu_clt,            // [NUM_CAMS] ->[TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
				num_textures,          // size_t            num_texture_tiles,  // number of texture tiles to process
				gpu_texture_indices,   // int             * gpu_texture_indices,// packed tile + bits (now only (1 << 7)
				gpu_port_offsets,      // float           * port_offsets,       // relative ports x,y offsets - just to scale differences, may be approximate
				texture_colors,        // int               colors,             // number of colors (3/1)
				(texture_colors == 1), // int               is_lwir,            // do not perform shot correction
				10.0,                  // float             min_shot,           // 10.0
				3.0,                   // float             scale_shot,         // 3.0
				1.5f,                  // float             diff_sigma,         // pixel value/pixel change
				10.0f,                 // float             diff_threshold,     // pixel value/pixel change
				3.0,                   // float             min_agree,          // minimal number of channels to agree on a point (real number to work with fuzzy averages)
				0.294118,              // float             weight0,            // scale for R
				0.117647,              // float             weight1,            // scale for B
				0.588235,              // float             weight2,            // scale for G
				1,                     // int               dust_remove,        // Do not reduce average weight when only one image differes much from the average
				keep_texture_weights,  // int               keep_weights,       // return channel weights after A in RGBA
    	// combining both non-overlap and overlap (each calculated if pointer is not null )
    			0, // const size_t      texture_rbg_stride, // in floats
    			(float *) 0, // float           * gpu_texture_rbg,     // (number of colors +1 + ?)*16*16 rgba texture tiles
				dstride_textures/sizeof(float), // const size_t      texture_stride,     // in floats (now 256*4 = 1024)
				gpu_textures);    // float           * gpu_texture_tiles);  // 4*16*16 rgba texture tiles
    	getLastCudaError("Kernel failure");
    	checkCudaErrors(hipDeviceSynchronize());
    	printf("test pass: %d\n",i);
    }
///	hipProfilerStop();
    sdkStopTimer(&timerTEXTURE);
    float avgTimeTEXTURES = (float)sdkGetTimerValue(&timerTEXTURE) / (float)numIterations;
    sdkDeleteTimer(&timerTEXTURE);
    printf("Average Texture run time =%f ms\n",  avgTimeTEXTURES);

    int rslt_texture_size =   num_textures * tile_texture_size;
    float * cpu_textures = (float *)malloc(rslt_texture_size * sizeof(float));



    checkCudaErrors(hipMemcpy2D(
    		cpu_textures,
			tile_texture_size * sizeof(float),
			gpu_textures,
			dstride_textures,
			tile_texture_size * sizeof(float),
			num_textures,
    		hipMemcpyDeviceToHost));

#ifndef NSAVE_TEXTURES
    		printf("Writing phase texture data to %s\n",  result_textures_file);
    		writeFloatsToFile(
    				cpu_textures,    // float *       data, // allocated array
					rslt_texture_size,    // int           size, // length in elements
					result_textures_file); // 			   const char *  path) // file path

//DBG_TILE
#ifdef DEBUG10
    		int texture_offset = DBG_TILE * tile_texture_size;
    		int chn = 0;
    		for (int i = 0; i < tile_texture_size; i++){
    			if ((i % 256) == 0){
    				printf("\nchn = %d\n", chn++);
    			}
    			printf("%10.4f", *(cpu_textures + texture_offset + i));
    			if (((i + 1) % 16) == 0){
    				printf("\n");
    			} else {
    				printf(" ");
    			}
    		}
//    int tile_texture_size = (texture_colors + 1 + (keep_texture_weights? (NUM_CAMS + texture_colors + 1): 0)) *256;
#endif // DEBUG9
#endif
    		free(cpu_textures);
#endif // ifndef NOTEXTURES


#define GEN_TEXTURE_LIST
#ifdef  GEN_TEXTURE_LIST
    		dim3 threads_list(1,1, 1); // TEXTURE_TILES_PER_BLOCK, 1);
    		dim3 grid_list   (1,1,1);
    		printf("threads_list=(%d, %d, %d)\n",threads_list.x,threads_list.y,threads_list.z);
    		printf("grid_list=(%d, %d, %d)\n",grid_list.x,grid_list.y,grid_list.z);
    		StopWatchInterface *timerTEXTURELIST = 0;
    		sdkCreateTimer(&timerTEXTURELIST);
    		for (int i = i0; i < numIterations; i++)
    		{
    			if (i == 0)
    			{
    				checkCudaErrors(hipDeviceSynchronize());
    				sdkResetTimer(&timerTEXTURELIST);
    				sdkStartTimer(&timerTEXTURELIST);
    			}

    			prepare_texture_list<<<grid_list,threads_list>>> (
    					gpu_tasks,             // struct tp_task   * gpu_tasks,
						tp_task_size,          // int                num_tiles,          // number of tiles in task list
						gpu_texture_indices,   // int              * gpu_texture_indices,// packed tile + bits (now only (1 << 7)
						gpu_num_texture_tiles, // int              * num_texture_tiles,  // number of texture tiles to process (8 elements)
						gpu_woi,               // int              * woi,                // x,y,width,height of the woi
						TILESX,                // int                width,  // <= TILESX, use for faster processing of LWIR images (should be actual + 1)
						TILESY);               // int                height); // <= TILESY, use for faster processing of LWIR images

    			getLastCudaError("Kernel failure");
    			checkCudaErrors(hipDeviceSynchronize());
    			printf("test pass: %d\n",i);
    		}
    		///	hipProfilerStop();
    		sdkStopTimer(&timerTEXTURELIST);
    		float avgTimeTEXTURESLIST = (float)sdkGetTimerValue(&timerTEXTURELIST) / (float)numIterations;
    		sdkDeleteTimer(&timerTEXTURELIST);
    		printf("Average TextureList run time =%f ms\n",  avgTimeTEXTURESLIST);

    		int cpu_num_texture_tiles[8];
    		checkCudaErrors(hipMemcpy(
    				cpu_woi,
					gpu_woi,
					4 * sizeof(float),
					hipMemcpyDeviceToHost));
    		printf("WOI x=%d, y=%d, width=%d, height=%d\n", cpu_woi[0], cpu_woi[1], cpu_woi[2], cpu_woi[3]);
    		checkCudaErrors(hipMemcpy(
    				cpu_num_texture_tiles,
					gpu_num_texture_tiles,
					8 * sizeof(float), // 8 sequences (0,2,4,6 - non-border, growing up;
					//1,3,5,7 - border, growing down from the end of the corresponding non-border buffers
					hipMemcpyDeviceToHost));
    		printf("cpu_num_texture_tiles=(%d(%d), %d(%d), %d(%d), %d(%d) -> %d tp_task_size=%d)\n",
    				cpu_num_texture_tiles[0], cpu_num_texture_tiles[1],
					cpu_num_texture_tiles[2], cpu_num_texture_tiles[3],
					cpu_num_texture_tiles[4], cpu_num_texture_tiles[5],
					cpu_num_texture_tiles[6], cpu_num_texture_tiles[7],
    				cpu_num_texture_tiles[0] + cpu_num_texture_tiles[1] +
					cpu_num_texture_tiles[2] + cpu_num_texture_tiles[3] +
					cpu_num_texture_tiles[4] + cpu_num_texture_tiles[5] +
					cpu_num_texture_tiles[6] + cpu_num_texture_tiles[7],
					tp_task_size
					);
    		for (int q = 0; q < 4; q++) {
    			checkCudaErrors(hipMemcpy(
    					texture_indices  + q * TILESX * (TILESYA >> 2),
						gpu_texture_indices  + q * TILESX * (TILESYA >> 2),
						cpu_num_texture_tiles[q] * sizeof(float), // change to cpu_num_texture_tiles when ready
						hipMemcpyDeviceToHost));
    		}
    		for (int q = 0; q < 4; q++) {
        		printf("%d: %3x:%3x %3x:%3x %3x:%3x %3x:%3x %3x:%3x %3x:%3x %3x:%3x %3x:%3x \n",q,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 0] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 0] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 1] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 1] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 2] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 2] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 3] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 3] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 4] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 4] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 5] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 5] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 6] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 6] >> 8) % TILESX,
        				(texture_indices[q * TILESX * (TILESYA >> 2) + 7] >> 8) / TILESX, (texture_indices[q * TILESX * (TILESYA >> 2) + 7] >> 8) % TILESX);
    		}
#endif //GEN_TEXTURE_LIST



#ifndef NOTEXTURE_RGBA
    dim3 threads_rgba(1, 1, 1);
    dim3 grid_rgba(1,1,1);
    printf("threads_rgba=(%d, %d, %d)\n", threads_rgba.x,threads_rgba.y,threads_rgba.z);
    printf("grid_rgba=(%d, %d, %d)\n",    grid_rgba.x,grid_rgba.y,grid_rgba.z);
    StopWatchInterface *timerRGBA = 0;
    sdkCreateTimer(&timerRGBA);

    for (int i = i0; i < numIterations; i++)
    {
    	if (i == 0)
    	{
    		checkCudaErrors(hipDeviceSynchronize());
    		sdkResetTimer(&timerRGBA);
    		sdkStartTimer(&timerRGBA);
    	}

    	generate_RBGA<<<grid_rgba,threads_rgba>>> (
    	// Parameters to generate texture tasks
                gpu_tasks,             // struct tp_task   * gpu_tasks,
                tp_task_size,          // int                num_tiles,          // number of tiles in task list
    	// declare arrays in device code?
	            gpu_texture_indices,   // int              * gpu_texture_indices,// packed tile + bits (now only (1 << 7)
	            gpu_num_texture_tiles, // int              * num_texture_tiles,  // number of texture tiles to process (8 elements)
	            gpu_woi,               // int              * woi,                // x,y,width,height of the woi
	            TILESX,                // int                width,  // <= TILESX, use for faster processing of LWIR images (should be actual + 1)
	            TILESY,                // int                height); // <= TILESY, use for faster processing of LWIR images
    	// Parameters for the texture generation
	            gpu_clt ,              // float          ** gpu_clt,            // [NUM_CAMS] ->[TILESY][TILESX][NUM_COLORS][DTT_SIZE*DTT_SIZE]
	            gpu_port_offsets,      // float           * port_offsets,       // relative ports x,y offsets - just to scale differences, may be approximate
	            texture_colors,        // int               colors,             // number of colors (3/1)
	            (texture_colors == 1), // int               is_lwir,            // do not perform shot correction
	            10.0,                  // float             min_shot,           // 10.0
	            3.0,                   // float             scale_shot,         // 3.0
	            1.5f,                  // float             diff_sigma,         // pixel value/pixel change
	            10.0f,                 // float             diff_threshold,     // pixel value/pixel change
	            3.0,                   // float             min_agree,          // minimal number of channels to agree on a point (real number to work with fuzzy averages)
	            0.294118,              // float             weight0,            // scale for R
	            0.117647,              // float             weight1,            // scale for B
	            0.588235,              // float             weight2,            // scale for G
	            1,                     // int               dust_remove,        // Do not reduce average weight when only one image differes much from the average
	            0,                     // int               keep_weights,       // return channel weights after A in RGBA
				dstride_textures_rbga/sizeof(float), // 	const size_t      texture_rbga_stride,     // in floats
				gpu_textures_rbga);    // 	float           * gpu_texture_tiles)    // (number of colors +1 + ?)*16*16 rgba texture tiles

    	getLastCudaError("Kernel failure");
    	checkCudaErrors(hipDeviceSynchronize());
    	printf("test pass: %d\n",i);
    }
    sdkStopTimer(&timerRGBA);
    float avgTimeRGBA = (float)sdkGetTimerValue(&timerRGBA) / (float)numIterations;
    sdkDeleteTimer(&timerRGBA);
    printf("Average Texture run time =%f ms\n",  avgTimeRGBA);

	checkCudaErrors(hipMemcpy(
			cpu_woi,
			gpu_woi,
			4 * sizeof(float),
			hipMemcpyDeviceToHost));
	printf("WOI x=%d, y=%d, width=%d, height=%d\n", cpu_woi[0], cpu_woi[1], cpu_woi[2], cpu_woi[3]);


	// temporarily use larger array (4 pixels each size, switch to hipMemcpy2DFromArray()
    int rgba_woi_width =  (cpu_woi[2] + 1) * DTT_SIZE;
    int rgba_woi_height = (cpu_woi[3] + 1)* DTT_SIZE;

    int rslt_rgba_size =     rgba_woi_width * rgba_woi_height * rbga_slices;
    float * cpu_textures_rgba = (float *)malloc(rslt_rgba_size * sizeof(float));

    checkCudaErrors(hipMemcpy2D(
    		cpu_textures_rgba,
			rgba_width * sizeof(float),
			gpu_textures_rbga,
			dstride_textures_rbga,
			rgba_width * sizeof(float),
			rgba_height * rbga_slices,
    		hipMemcpyDeviceToHost));

#ifndef NSAVE_TEXTURES
    printf("Writing RBGA texture slices to %s\n",  result_textures_rgba_file);
    writeFloatsToFile(
    		cpu_textures_rgba,    // float *       data, // allocated array
			rslt_rgba_size,    // int           size, // length in elements
			result_textures_rgba_file); // 			   const char *  path) // file path
#endif
#ifdef DEBUG11
    int rgba_offset = (DBG_TILE_Y - cpu_woi[1]) * DTT_SIZE * rgba_woi_width  + (DBG_TILE_X - cpu_woi[0]);
    for (int chn = 0; chn < rbga_slices; chn++){
    	printf("\nchn = %d\n", chn);
    	int rgba_offset_chn = rgba_offset + chn * rgba_woi_width * rgba_woi_height;

    	for (int i = 0; i < 8; i++){
    		for (int j = 0; j < 8; j++){
    			printf("%10.4f ", *(cpu_textures_rgba + rgba_offset_chn + i * rgba_woi_width + j));
    		}
    		printf("\n");
    	}
    }
#endif // DEBUG11
    free(cpu_textures_rgba);
#endif // ifndef NOTEXTURES














#ifdef SAVE_CLT
    free(cpu_clt);
#endif

    free (host_kern_buf);
    // TODO: move somewhere when all is done
    for (int ncam = 0; ncam < NUM_CAMS; ncam++) {
    	checkCudaErrors(hipFree(gpu_kernels_h[ncam]));
    	checkCudaErrors(hipFree(gpu_kernel_offsets_h[ncam]));
    	checkCudaErrors(hipFree(gpu_images_h[ncam]));
    	checkCudaErrors(hipFree(gpu_clt_h[ncam]));
#ifndef NOICLT
    	checkCudaErrors(hipFree(gpu_corr_images_h[ncam]));
#endif
    }
	checkCudaErrors(hipFree(gpu_tasks));
	checkCudaErrors(hipFree(gpu_kernels));
	checkCudaErrors(hipFree(gpu_kernel_offsets));
	checkCudaErrors(hipFree(gpu_images));
	checkCudaErrors(hipFree(gpu_clt));
//	checkCudaErrors(hipFree(gpu_corr_images));
	checkCudaErrors(hipFree(gpu_corrs));
	checkCudaErrors(hipFree(gpu_corr_indices));
	checkCudaErrors(hipFree(gpu_texture_indices));
	checkCudaErrors(hipFree(gpu_port_offsets));
	checkCudaErrors(hipFree(gpu_textures));
	checkCudaErrors(hipFree(gpu_textures_rbga));
	checkCudaErrors(hipFree(gpu_woi));
	checkCudaErrors(hipFree(gpu_num_texture_tiles));



	exit(0);
}

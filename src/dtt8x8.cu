#include "hip/hip_runtime.h"
/**
 **
 ** dtt8x8.cuh
 **
 ** Copyright (C) 2018 Elphel, Inc.
 **
 ** -----------------------------------------------------------------------------**
 **
 **  dtt8x8.cuh is free software: you can redistribute it and/or modify
 **  it under the terms of the GNU General Public License as published by
 **  the Free Software Foundation, either version 3 of the License, or
 **  (at your option) any later version.
 **
 **  This program is distributed in the hope that it will be useful,
 **  but WITHOUT ANY WARRANTY; without even the implied warranty of
 **  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 **  GNU General Public License for more details.
 **
 **  You should have received a copy of the GNU General Public License
 **  along with this program.  If not, see <http://www.gnu.org/licenses/>.
 **
 **  Additional permission under GNU GPL version 3 section 7
 **
 **  If you modify this Program, or any covered work, by linking or
 **  combining it with NVIDIA Corporation's CUDA libraries from the
 **  NVIDIA CUDA Toolkit (or a modified version of those libraries),
 **  containing parts covered by the terms of NVIDIA CUDA Toolkit
 **  EULA, the licensors of this Program grant you additional
 **  permission to convey the resulting work.
 ** -----------------------------------------------------------------------------**
 */

/**
**************************************************************************
* \file dtt8x8.cuh
* \brief DCT-II, DST-II, DCT-IV and DST-IV for Complex Lapped Transform of 16x16 (stride 8)
*        in GPU
* This file contains building blocks for the 16x16 stride 8 COmplex Lapped Transform (CLT)
* implementation. DTT-IV are used for forward and inverse 2D CLT, DTT-II - to convert correlation
* results from the frequency to pixel domain. DTT-III (inverse of DTT-II) is not implemented
* here it is used to convert convolution kernels and LPF to the frequency domain - done in
* software.
*
* This file is cpompatible with both runtime and driver API, runtime is used for development
* with Nvidia Nsight, driver API when calling these kernels from Java
*/
#ifndef JCUDA
#include "dtt8x8.h"
#endif

//#define HIP_INF_F            __int_as_float(0x7f800000)
/*
 Python code to generate constant coefficients:
def dct_constants():
    COSPI_1_8_SQRT2 = math.cos(math.pi/8)*math.sqrt(2.0)
    COSPI_3_8_SQRT2 = math.cos(3*math.pi/8)*math.sqrt(2.0)
    SQRT_2 = math.sqrt(2.0)
    SQRT1_2 = 1/math.sqrt(2.0)
    SQRT1_8 = 1/math.sqrt(8.0)
    CN = [[math.cos((2*k+1)*(math.pi/(8*(2 << t))))  for k in range (2 << t)] for t in range (2)]
    SN = [[math.sin((2*k+1)*(math.pi/(8*(2 << t))))  for k in range (2 << t)] for t in range (2)]
    print("__constant__ float COSPI_1_8_SQRT2 = %ff;"%(COSPI_1_8_SQRT2))
    print("__constant__ float COSPI_3_8_SQRT2 = %ff;"%(COSPI_3_8_SQRT2))
    print("__constant__ float SQRT_2 = %ff;"%         (SQRT_2))
    print("__constant__ float SQRT1_2 = %ff;"%        (SQRT1_2))
    print("__constant__ float SQRT1_8 = %ff;"%        (SQRT1_8))
    print("__constant__ float COSN1[] = {%ff,%ff};"%         (CN[0][0],CN[0][1]))
    print("__constant__ float COSN2[] = {%ff,%ff,%ff,%ff};"% (CN[1][0],CN[1][1],CN[1][2],CN[1][3]))
    print("__constant__ float SINN1[] = {%ff,%ff};"%         (SN[0][0],SN[0][1]))
    print("__constant__ float SINN2[] = {%ff,%ff,%ff,%ff};"% (SN[1][0],SN[1][1],SN[1][2],SN[1][3]))
*/
__constant__ float COSPI_1_8_SQRT2 = 1.306563f;
__constant__ float COSPI_3_8_SQRT2 = 0.541196f;
__constant__ float SQRT_2 = 1.414214f;
__constant__ float SQRT1_2 = 0.707107f;
__constant__ float SQRT1_8 = 0.353553f;
__constant__ float COSN1[] = {0.980785f,0.831470f};
__constant__ float COSN2[] = {0.995185f,0.956940f,0.881921f,0.773010f};
__constant__ float SINN1[] = {0.195090f,0.555570f};
__constant__ float SINN2[] = {0.098017f,0.290285f,0.471397f,0.634393f};
__constant__ int imclt_indx9[16] = {0x28,0x29,0x2a,0x2b,0x2b,0x2a,0x29,0x28,0x27,0x26,0x25,0x24,0x24,0x25,0x26,0x27};
__constant__ float idct_signs[4][4][4] ={
		{ // quadrant 0, each elements corresponds to 4x4 pixel output, covering altogether 16x16
				{ 1,-1,-1,-1},
				{-1, 1, 1, 1},
				{-1, 1, 1, 1},
				{-1, 1, 1, 1}
		},{ // quadrant 1, each elements corresponds to 4x4 pixel output, covering altogether 16x16
				{ 1, 1, 1,-1},
				{-1,-1,-1, 1},
				{-1,-1,-1, 1},
				{-1,-1,-1, 1}
		},{ // quadrant 2, each elements corresponds to 4x4 pixel output, covering altogether 16x16
				{ 1,-1,-1,-1},
				{ 1,-1,-1,-1},
				{ 1,-1,-1,-1},
				{-1, 1, 1, 1}
		},{ // quadrant 3, each elements corresponds to 4x4 pixel output, covering altogether 16x16
				{ 1, 1, 1,-1},
				{ 1, 1, 1,-1},
				{ 1, 1, 1,-1},
				{-1,-1,-1, 1}
		}};
__constant__ float HWINDOW2[] =  {0.049009f, 0.145142f, 0.235698f, 0.317197f,
                                  0.386505f, 0.440961f, 0.478470f, 0.497592f};


inline __device__ void dttii_shared_mem_nonortho(float * x0,  int inc, int dst_not_dct); // does not scale by y[0] (y[7]) by 1/sqrt[0]
inline __device__ void dttii_shared_mem(float * x0,  int inc, int dst_not_dct);   // used in GPU_DTT24_DRV
inline __device__ void dttiv_shared_mem(float * x0,  int inc, int dst_not_dct);   // used in GPU_DTT24_DRV
inline __device__ void dttiv_nodiverg  (float * x,   int inc, int dst_not_dct);   // not used
inline __device__ void dctiv_nodiverg  (float * x0,  int inc);                    // used in TP
inline __device__ void dstiv_nodiverg  (float * x0,  int inc);                    // used in TP

inline __device__ void dct_ii8         ( float x[8], float y[8]); // x,y point to 8-element arrays each // not used
inline __device__ void dct_iv8         ( float x[8], float y[8]); // x,y point to 8-element arrays each // not used
inline __device__ void dst_iv8         ( float x[8], float y[8]); // x,y point to 8-element arrays each // not used
inline __device__ void _dctii_nrecurs8 ( float x[8], float y[8]); // x,y point to 8-element arrays each // not used
inline __device__ void _dctiv_nrecurs8 ( float x[8], float y[8]); // x,y point to 8-element arrays each // not used


/**
**************************************************************************
*  Converts 2D image (in the GPU memory) using 8x8 DTT 8x8 tiles.
*  Mostly for testing and profiling individual conversions
*
* \param dst                        [OUT] - Coefficients as 8x8 tiles
* \param src                         [IN] - Source image of floats
* \param src_stride                  [IN] - Source image stride
* \param mode                        [IN] - DTT mode:
*     0 - horizontal DCT-IV followed by vertical DCT-IV
*     1 - horizontal DST-IV followed by vertical DCT-IV
*     2 - horizontal DCT-IV followed by vertical DST-IV
*     3 - horizontal DST-IV followed by vertical DST-IV
*     4 - horizontal DCT-II followed by vertical DCT-II
*     5 - horizontal DST-II followed by vertical DCT-II
*     6 - horizontal DCT-II followed by vertical DST-II
*     7 - horizontal DST-II followed by vertical DST-II
*
* \return None
*/
#ifdef BBBB
extern "C"
__global__ void GPU_DTT24_DRV(float *dst, float *src, int src_stride, int dtt_mode)
{
	int dtt_mode0 = dtt_mode & 1;
	int dtt_mode1 = (dtt_mode >>1) & 1;

    __shared__ float block[DTTTEST_BLOCK_HEIGHT * DTTTEST_BLK_STRIDE];

    int OffsThreadInRow = threadIdx.y * DTT_SIZE + threadIdx.x;
    int OffsThreadInCol = threadIdx.z * DTT_SIZE;
    src += ((blockIdx.y * DTTTEST_BLOCK_HEIGHT + OffsThreadInCol) * src_stride) + blockIdx.x * DTTTEST_BLOCK_WIDTH + OffsThreadInRow;
    dst += ((blockIdx.y * DTTTEST_BLOCK_HEIGHT + OffsThreadInCol) * src_stride) + blockIdx.x * DTTTEST_BLOCK_WIDTH + OffsThreadInRow;
    float *bl_ptr = block + OffsThreadInCol * DTTTEST_BLK_STRIDE + OffsThreadInRow;

#pragma unroll

    for (unsigned int i = 0; i < DTT_SIZE; i++)
        bl_ptr[i * DTTTEST_BLK_STRIDE] = src[i * src_stride];

    __syncthreads();
    // horizontal pass
    if (dtt_mode > 3) {
    	dttii_shared_mem                   (block + (OffsThreadInCol + threadIdx.x) * DTTTEST_BLK_STRIDE + OffsThreadInRow - threadIdx.x, 1, dtt_mode0);
    } else {
    	dttiv_shared_mem                   (block + (OffsThreadInCol + threadIdx.x) * DTTTEST_BLK_STRIDE + OffsThreadInRow - threadIdx.x, 1, dtt_mode0);
    }

    __syncthreads();
    // vertical pass
    if (dtt_mode > 3) {
    	dttii_shared_mem                    (bl_ptr, DTTTEST_BLK_STRIDE, dtt_mode1);
    } else {
    	dttiv_shared_mem                    (bl_ptr, DTTTEST_BLK_STRIDE, dtt_mode1);
    }
    __syncthreads();
    for (unsigned int i = 0; i < DTT_SIZE; i++)
        dst[i * src_stride] = bl_ptr[i * DTTTEST_BLK_STRIDE];
}
#endif //#ifdef BBBB



inline __device__ void _dctiv_nrecurs8( float x[8], float y[8]) // x,y point to 8-element arrays each
{
	float u00=            ( COSN2[0] * x[0] + SINN2[0] * x[7]);
	float u10=            (-SINN2[3] * x[3] + COSN2[3] * x[4]);

	float u01=            ( COSN2[1] * x[1] + SINN2[1] * x[6]);
	float u11=           -(-SINN2[2] * x[2] + COSN2[2] * x[5]);

	float u02=            ( COSN2[2] * x[2] + SINN2[2] * x[5]);
	float u12=            (-SINN2[1] * x[1] + COSN2[1] * x[6]);

	float u03=            ( COSN2[3] * x[3] + SINN2[3] * x[4]);
	float u13=           -(-SINN2[0] * x[0] + COSN2[0] * x[7]);

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;


	y[0] =  SQRT_2 * v00;    // w0[0];
	y[1] =  v01 -  vb11;    // w1[0];
	// j == 1
	y[2] =  v01 +  vb11;    // w0[1];
	y[3] =  v02 +  vb01;    // w1[1];
	// j == 2
	y[4] =  v02 -  vb01;    // w0[2];
	y[5] =  v03 -  vb10;    // w1[2]; - same as y[3]
	// j == 3
	y[6] =  v03 +  vb10;    // w0[3];
	y[7] =  SQRT_2 * vb00;    // w1[3];
}

__device__ void _dttiv(float x0, float x1,float x2, float x3,float x4, float x5,float x6, float x7,
		float *y0, float *y1, float *y2, float *y3, float *y4, float *y5, float *y6, float *y7, int dst_not_dct)
{
	float u00, u01, u02, u03, u10, u11, u12, u13;
	if (dst_not_dct) { // DSTIV
		u00=  ( COSN2[0] * x7 + SINN2[0] * x0);
		u10=  (-SINN2[3] * x4 + COSN2[3] * x3);

		u01=  ( COSN2[1] * x6 + SINN2[1] * x1);
		u11= -(-SINN2[2] * x5 + COSN2[2] * x2);

		u02=  ( COSN2[2] * x5 + SINN2[2] * x2);
		u12=  (-SINN2[1] * x6 + COSN2[1] * x1);

		u03=  ( COSN2[3] * x4 + SINN2[3] * x3);
		u13= -(-SINN2[0] * x7 + COSN2[0] * x0);
	} else { // DCTIV
		u00=  ( COSN2[0] * x0 + SINN2[0] * x7);
		u10=  (-SINN2[3] * x3 + COSN2[3] * x4);

		u01=  ( COSN2[1] * x1 + SINN2[1] * x6);
		u11= -(-SINN2[2] * x2 + COSN2[2] * x5);

		u02=  ( COSN2[2] * x2 + SINN2[2] * x5);
		u12=  (-SINN2[1] * x1 + COSN2[1] * x6);

		u03=  ( COSN2[3] * x3 + SINN2[3] * x4);
		u13= -(-SINN2[0] * x0 + COSN2[0] * x7);
	}

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;

	*y0 =  v00 * 0.5f;              // w0[0];
	// j == 1
	*y2 =  (v01 +  vb11) * SQRT1_8; // w0[1];
	// j == 2
	*y4 =  (v02 -  vb01) * SQRT1_8; // w0[2];
	// j == 3
	*y6 =  (v03 +  vb10) * SQRT1_8; // w0[3];
	if (dst_not_dct) { // DSTIV
		*y1 =  (vb11 - v01)  * SQRT1_8; // w1[0];
		*y3 = -(v02 +  vb01) * SQRT1_8; // w1[1];
		*y5 =  (vb10 - v03)  * SQRT1_8; // w1[2]; - same as y[3]
		*y7 = -vb00 * 0.5f;             // w1[3];
	} else {
		*y1 =  (v01 -  vb11) * SQRT1_8; // w1[0];
		*y3 =  (v02 +  vb01) * SQRT1_8; // w1[1];
		*y5 =  (v03 -  vb10) * SQRT1_8; // w1[2]; - same as y[3]
		*y7 =  vb00 * 0.5f;             // w1[3];
	}
}

inline __device__ void dttii_shared_mem(float * x0,  int inc, int dst_not_dct)
{
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	if (dst_not_dct) { // DSTII
		// invert odd input samples
		u00= ( (*x0) - (*x7));
		u10= ( (*x0) + (*x7));

		u01= (-(*x1) + (*x6));
		u11= (-(*x1) - (*x6));

		u02= ( (*x2) - (*x5));
		u12= ( (*x2) + (*x5));

		u03= (-(*x3) + (*x4));
		u13= (-(*x3) - (*x4));
	} else { // DCTII
		u00= ( (*x0) + (*x7));
		u10= ( (*x0) - (*x7));

		u01= ( (*x1) + (*x6));
		u11= ( (*x1) - (*x6));

		u02= ( (*x2) + (*x5));
		u12= ( (*x2) - (*x5));

		u03= ( (*x3) + (*x4));
		u13= ( (*x3) - (*x4));
	}
	//	_dctii_nrecurs4(u00,u01, u02, u03, &v00, &v01, &v02, &v03);

		float w00= u00 + u03;
		float w10= u00 - u03;

		float w01= (u01 + u02);
		float w11= (u01 - u02);

		float v01= COSPI_1_8_SQRT2 * w10 + COSPI_3_8_SQRT2 * w11;
		float v03= COSPI_3_8_SQRT2 * w10 - COSPI_1_8_SQRT2 * w11;
	//	_dctiv_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);
		float w20=            ( COSN1[0] * u10 + SINN1[0] * u13);
		float w30=            (-SINN1[1] * u11 + COSN1[1] * u12);

		float w21=            ( COSN1[1] * u11 + SINN1[1] * u12);
		float w31=           -(-SINN1[0] * u10 + COSN1[0] * u13);
		float v11 = w20 - w21 - w30 + w31;
		float v12 = w20 - w21 + w30 - w31;

	if (dst_not_dct) { // DSTII
		// Invert output sequence
		*x0 =   (w30 + w31)*  0.5f;    // v13 * SQRT1_8; z10 * 0.5f
		*x1 =   v03 *         SQRT1_8;

		*x2 =   v12 *         SQRT1_8;
		*x3 =   (w00 - w01) * SQRT1_8; // v02 * SQRT1_8

		*x4 =   v11 *         SQRT1_8;
		*x5 =   v01 *         SQRT1_8;

		*x6 =   (w20 + w21) * 0.5f;    // v10 * SQRT1_8; z00 * 0.5f;
		*x7 =   (w00 + w01) * SQRT1_8; // v00 * SQRT1_8
	} else {
		*x0 =   (w00 + w01) * SQRT1_8; // v00 * SQRT1_8
		*x1 =   (w20 + w21) * 0.5f;    // v10 * SQRT1_8; z00 * 0.5f;

		*x2 =   v01 *         SQRT1_8;
		*x3 =   v11 *         SQRT1_8;

		*x4 =   (w00 - w01) * SQRT1_8; // v02 * SQRT1_8
		*x5 =   v12 *         SQRT1_8;

		*x6 =   v03 *         SQRT1_8;
		*x7 =   (w30 + w31)*  0.5f;    // v13 * SQRT1_8; z10 * 0.5f
	}
}

inline __device__ void dttii_shared_mem_nonortho(float * x0,  int inc, int dst_not_dct)
{
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	if (dst_not_dct) { // DSTII
		// invert odd input samples
		u00= ( (*x0) - (*x7));
		u10= ( (*x0) + (*x7));

		u01= (-(*x1) + (*x6));
		u11= (-(*x1) - (*x6));

		u02= ( (*x2) - (*x5));
		u12= ( (*x2) + (*x5));

		u03= (-(*x3) + (*x4));
		u13= (-(*x3) - (*x4));
	} else { // DCTII
		u00= ( (*x0) + (*x7));
		u10= ( (*x0) - (*x7));

		u01= ( (*x1) + (*x6));
		u11= ( (*x1) - (*x6));

		u02= ( (*x2) + (*x5));
		u12= ( (*x2) - (*x5));

		u03= ( (*x3) + (*x4));
		u13= ( (*x3) - (*x4));
	}
	//	_dctii_nrecurs4(u00,u01, u02, u03, &v00, &v01, &v02, &v03);

		float w00= u00 + u03;
		float w10= u00 - u03;

		float w01= (u01 + u02);
		float w11= (u01 - u02);

		float v01= COSPI_1_8_SQRT2 * w10 + COSPI_3_8_SQRT2 * w11;
		float v03= COSPI_3_8_SQRT2 * w10 - COSPI_1_8_SQRT2 * w11;
	//	_dctiv_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);
		float w20=            ( COSN1[0] * u10 + SINN1[0] * u13);
		float w30=            (-SINN1[1] * u11 + COSN1[1] * u12);

		float w21=            ( COSN1[1] * u11 + SINN1[1] * u12);
		float w31=           -(-SINN1[0] * u10 + COSN1[0] * u13);
		float v11 = w20 - w21 - w30 + w31;
		float v12 = w20 - w21 + w30 - w31;

	if (dst_not_dct) { // DSTII
		// Invert output sequence
		*x0 =   (w30 + w31)*  0.5f;    // v13 * SQRT1_8; z10 * 0.5f
		*x1 =   v03 *         SQRT1_8;

		*x2 =   v12 *         SQRT1_8;
		*x3 =   (w00 - w01) * SQRT1_8; // v02 * SQRT1_8

		*x4 =   v11 *         SQRT1_8;
		*x5 =   v01 *         SQRT1_8;

		*x6 =   (w20 + w21) * 0.5f;    // v10 * SQRT1_8; z00 * 0.5f;
		*x7 =   (w00 + w01) * 0.5f;    // SQRT1_8; // v00 * SQRT1_8 //*** no 1/sqrt(2)!
	} else {
		*x0 =   (w00 + w01) * 0.5f;    // SQRT1_8; // v00 * SQRT1_8 //*** no 1/sqrt(2)!
		*x1 =   (w20 + w21) * 0.5f;    // v10 * SQRT1_8; z00 * 0.5f;

		*x2 =   v01 *         SQRT1_8;
		*x3 =   v11 *         SQRT1_8;

		*x4 =   (w00 - w01) * SQRT1_8; // v02 * SQRT1_8
		*x5 =   v12 *         SQRT1_8;

		*x6 =   v03 *         SQRT1_8;
		*x7 =   (w30 + w31)*  0.5f;    // v13 * SQRT1_8; z10 * 0.5f
	}
}

inline __device__ void dttiv_shared_mem(float * x0,  int inc, int dst_not_dct)
{
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	if (dst_not_dct) { // DSTIV
		u00=  ( COSN2[0] * (*x7) + SINN2[0] * (*x0));
		u10=  (-SINN2[3] * (*x4) + COSN2[3] * (*x3));

		u01=  ( COSN2[1] * (*x6) + SINN2[1] * (*x1));
		u11= -(-SINN2[2] * (*x5) + COSN2[2] * (*x2));

		u02=  ( COSN2[2] * (*x5) + SINN2[2] * (*x2));
		u12=  (-SINN2[1] * (*x6) + COSN2[1] * (*x1));

		u03=  ( COSN2[3] * (*x4) + SINN2[3] * (*x3));
		u13= -(-SINN2[0] * (*x7) + COSN2[0] * (*x0));
	} else { // DCTIV
		u00=  ( COSN2[0] * (*x0) + SINN2[0] * (*x7));
		u10=  (-SINN2[3] * (*x3) + COSN2[3] * (*x4));

		u01=  ( COSN2[1] * (*x1) + SINN2[1] * (*x6));
		u11= -(-SINN2[2] * (*x2) + COSN2[2] * (*x5));

		u02=  ( COSN2[2] * (*x2) + SINN2[2] * (*x5));
		u12=  (-SINN2[1] * (*x1) + COSN2[1] * (*x6));

		u03=  ( COSN2[3] * (*x3) + SINN2[3] * (*x4));
		u13= -(-SINN2[0] * (*x0) + COSN2[0] * (*x7));
	}

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;


	*x0 =  v00 * 0.5f;              // w0[0];
	*x2 =  (v01 +  vb11) * SQRT1_8; // w0[1];
	*x4 =  (v02 -  vb01) * SQRT1_8; // w0[2];
	*x6 =  (v03 +  vb10) * SQRT1_8; // w0[3];
	if (dst_not_dct) { // DSTIV
		*x1 =  (vb11 - v01)  * SQRT1_8; // w1[0];
		*x3 = -(v02 +  vb01) * SQRT1_8; // w1[1];
		*x5 =  (vb10 - v03)  * SQRT1_8; // w1[2]; - same as y[3]
		*x7 = -vb00 * 0.5f;             // w1[3];
	} else {
		*x1 =  (v01 -  vb11) * SQRT1_8; // w1[0];
		*x3 =  (v02 +  vb01) * SQRT1_8; // w1[1];
		*x5 =  (v03 -  vb10) * SQRT1_8; // w1[2]; - same as y[3]
		*x7 =  vb00 * 0.5f;             // w1[3];
	}
}

inline __device__ void dttiv_nodiverg(float * x,  int inc, int dst_not_dct)
{
	float sgn = 1 - 2* dst_not_dct;
	float *y0 = x;
	float *y1 = y0 + inc;
	float *y2 = y1 + inc;
	float *y3 = y2 + inc;
	float *y4 = y3 + inc;
	float *y5 = y4 + inc;
	float *y6 = y5 + inc;
	float *y7 = y6 + inc;

	float *x0 =  x + dst_not_dct * 7 * inc;
	// negate inc, replace
	inc *= sgn;
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	u00=  ( COSN2[0] * (*x0) + SINN2[0] * (*x7));
	u10=  (-SINN2[3] * (*x3) + COSN2[3] * (*x4));

	u01=  ( COSN2[1] * (*x1) + SINN2[1] * (*x6));
	u11= -(-SINN2[2] * (*x2) + COSN2[2] * (*x5));

	u02=  ( COSN2[2] * (*x2) + SINN2[2] * (*x5));
	u12=  (-SINN2[1] * (*x1) + COSN2[1] * (*x6));

	u03=  ( COSN2[3] * (*x3) + SINN2[3] * (*x4));
	u13= -(-SINN2[0] * (*x0) + COSN2[0] * (*x7));

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;


	*y0 =  v00 * 0.5f;              // w0[0];
	*y2 =  (v01 +  vb11) * SQRT1_8; // w0[1];
	*y4 =  (v02 -  vb01) * SQRT1_8; // w0[2];
	*y6 =  (v03 +  vb10) * SQRT1_8; // w0[3];
	*y1 =  sgn * (v01 -  vb11) * SQRT1_8; // w1[0];
	*y3 =  sgn * (v02 +  vb01) * SQRT1_8; // w1[1];
	*y5 =  sgn * (v03 -  vb10) * SQRT1_8; // w1[2]; - same as y[3]
	*y7 =  sgn * vb00 * 0.5f;             // w1[3];
}

inline __device__ void dctiv_nodiverg(float * x0,  int inc)
{
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	u00=  ( COSN2[0] * (*x0) + SINN2[0] * (*x7));
	u10=  (-SINN2[3] * (*x3) + COSN2[3] * (*x4));

	u01=  ( COSN2[1] * (*x1) + SINN2[1] * (*x6));
	u11= -(-SINN2[2] * (*x2) + COSN2[2] * (*x5));

	u02=  ( COSN2[2] * (*x2) + SINN2[2] * (*x5));
	u12=  (-SINN2[1] * (*x1) + COSN2[1] * (*x6));

	u03=  ( COSN2[3] * (*x3) + SINN2[3] * (*x4));
	u13= -(-SINN2[0] * (*x0) + COSN2[0] * (*x7));

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;


	*x0 =  v00 * 0.5f;              // w0[0];
	*x2 =  (v01 +  vb11) * SQRT1_8; // w0[1];
	*x4 =  (v02 -  vb01) * SQRT1_8; // w0[2];
	*x6 =  (v03 +  vb10) * SQRT1_8; // w0[3];
	*x1 =  (v01 -  vb11) * SQRT1_8; // w1[0];
	*x3 =  (v02 +  vb01) * SQRT1_8; // w1[1];
	*x5 =  (v03 -  vb10) * SQRT1_8; // w1[2]; - same as y[3]
	*x7 =   vb00 * 0.5f;             // w1[3];
}

inline __device__ void dstiv_nodiverg(float * x,  int inc)
{
	float *x0 =  x +  7 * inc;
	// negate inc, replace
	inc = -inc;
	float *x1 = x0 + inc;
	float *x2 = x1 + inc;
	float *x3 = x2 + inc;
	float *x4 = x3 + inc;
	float *x5 = x4 + inc;
	float *x6 = x5 + inc;
	float *x7 = x6 + inc;
	float u00, u01, u02, u03, u10, u11, u12, u13;
	u00=  ( COSN2[0] * (*x0) + SINN2[0] * (*x7));
	u10=  (-SINN2[3] * (*x3) + COSN2[3] * (*x4));

	u01=  ( COSN2[1] * (*x1) + SINN2[1] * (*x6));
	u11= -(-SINN2[2] * (*x2) + COSN2[2] * (*x5));

	u02=  ( COSN2[2] * (*x2) + SINN2[2] * (*x5));
	u12=  (-SINN2[1] * (*x1) + COSN2[1] * (*x6));

	u03=  ( COSN2[3] * (*x3) + SINN2[3] * (*x4));
	u13= -(-SINN2[0] * (*x0) + COSN2[0] * (*x7));

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float ua00= u00 + u03;
	float ua10= u00 - u03;

	float ua01= u01 + u02;
	float ua11= u01 - u02;

	float v00= ua00 + ua01;
	float v02= ua00 - ua01;

	float v01= COSPI_1_8_SQRT2 * ua10 + COSPI_3_8_SQRT2 * ua11;
	float v03= COSPI_3_8_SQRT2 * ua10 - COSPI_1_8_SQRT2 * ua11;

//	_dctii_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);

	float ub00= u10 + u13;
	float ub10= u10 - u13;

	float ub01= u11 + u12;
	float ub11= u11 - u12;

	float vb00= ub00 + ub01;
	float vb01= ub00 - ub01;

	float vb10= COSPI_1_8_SQRT2*ub10 + COSPI_3_8_SQRT2*ub11;
	float vb11= COSPI_3_8_SQRT2*ub10 - COSPI_1_8_SQRT2*ub11;


	*x7 =  v00 * 0.5f;              // w0[0];
	*x5 =  (v01 +  vb11) * SQRT1_8; // w0[1];
	*x3 =  (v02 -  vb01) * SQRT1_8; // w0[2];
	*x1 =  (v03 +  vb10) * SQRT1_8; // w0[3];

	*x6 =  (vb11 - v01)  * SQRT1_8; // w1[0];
	*x4 = -(v02 +  vb01) * SQRT1_8; // w1[1];
	*x2 =  (vb10 - v03)  * SQRT1_8; // w1[2]; - same as y[3]
	*x0 = -vb00 * 0.5f;             // w1[3];
}



inline  __device__ void _dctii_nrecurs8( float x[8], float y[8]) // x,y point to 8-element arrays each
{
	float u00= (x[0] + x[7]);
	float u10= (x[0] - x[7]);

	float u01= (x[1] + x[6]);
	float u11= (x[1] - x[6]);

	float u02= (x[2] + x[5]);
	float u12= (x[2] - x[5]);

	float u03= (x[3] + x[4]);
	float u13= (x[3] - x[4]);

//	_dctii_nrecurs4(u00, u01, u02, u03, &v00, &v01, &v02, &v03);

	float w00= u00 + u03;
	float w10= u00 - u03;

	float w01= (u01 + u02);
	float w11= (u01 - u02);

	float v00= w00 + w01;
	float v02= w00 - w01;
	float v01= COSPI_1_8_SQRT2 * w10 + COSPI_3_8_SQRT2 * w11;
	float v03= COSPI_3_8_SQRT2 * w10 - COSPI_1_8_SQRT2 * w11;

//	_dctiv_nrecurs4(u10, u11, u12, u13, &v10, &v11, &v12, &v13);
	float w20=            ( COSN1[0] * u10 + SINN1[0] * u13);
	float w30=            (-SINN1[1] * u11 + COSN1[1] * u12);

	float w21=            ( COSN1[1] * u11 + SINN1[1] * u12);
	float w31=           -(-SINN1[0] * u10 + COSN1[0] * u13);

//	_dctii_nrecurs2(u00, u01, &v00, &v01);
	float z00= w20 + w21;
	float z01= w20 - w21;

//	_dctii_nrecurs2(u10, u11, &v10, &v11);
	float z10= w30 + w31;
	float z11= w30 - w31;

	float v10 = SQRT_2 * z00;
	float v11 = z01 - z11;

	float v12 = z01 + z11;
	float v13 = SQRT_2 * z10;

	y[0] =   v00;
	y[1] =   v10;

	y[2] =   v01;
	y[3] =   v11;

	y[4] =   v02;
	y[5] =   v12;

	y[6] =   v03;
	y[7] =   v13;
}

inline  __device__ void dct_ii8( float x[8], float y[8]) // x,y point to 8-element arrays each
{
	_dctii_nrecurs8(x, y);
#pragma unroll
	for (int i = 0; i < 8 ; i++) {
		y[i] *= SQRT1_8;
	}
}


__device__ void dct_iv8( float x[8], float y[8]) // x,y point to 8-element arrays each
{
	_dctiv_nrecurs8(x, y);
#pragma unroll
	for (int i = 0; i < 8 ; i++) {
		y[i] *= SQRT1_8;
	}

}

inline __device__ void dst_iv8( float x[8], float y[8]) // x,y point to 8-element arrays each
{
	float xr[8];
#pragma unroll
	for (int i=0; i < 8;i++){
		xr[i] = x[7 - i];
	}
	_dctiv_nrecurs8(xr, y);
#pragma unroll
	for (int i=0; i < 8;i+=2){
		y[i]   *=  SQRT1_8;
		y[i+1] *= -SQRT1_8;
	}
}


//=========================== 2D functions ===============
__device__ void corrUnfoldTile(
		int corr_radius,
		float* qdata0, //    [4][DTT_SIZE][DTT_SIZE1], // 4 quadrants of the clt data, rows extended to optimize shared ports
		float* rslt)  //   [DTT_SIZE2M1][DTT_SIZE2M1]) // 15x15
{
	int size2r1 = 2 * corr_radius + 1; // 15
	int crp1 = corr_radius + 1;        //8
///	const int rslt_base_index = DTT_SIZE2M1 * (DTT_SIZE) - DTT_SIZE; // offset of the center
	int rslt_base_index = size2r1 * crp1 - crp1; // offset of the center

	float * qdata1 = qdata0 + (DTT_SIZE * DTT_SIZE1);
	float * qdata2 = qdata1 + (DTT_SIZE * DTT_SIZE1);
	float * qdata3 = qdata2 + (DTT_SIZE * DTT_SIZE1);
	int i = threadIdx.x;
	if (i > corr_radius) {
		return; // not needed, only use inner
	}
//	printf("\corrUnfoldTile() corr_radius=%d, i=%d\n",corr_radius,i);
	float corr_pixscale = 0.25f;
	int i_transform_size = i * DTT_SIZE1; // used to address source rows which are 9 long
	int im1_transform_size = i_transform_size - DTT_SIZE1; // negative for i = 0, use only after divergence
///	int rslt_row_offs = i * DTT_SIZE2M1;
	int rslt_row_offs = i * size2r1;
	int rslt_base_index_p = rslt_base_index + rslt_row_offs; // i * DTT_SIZE2M1;
	int rslt_base_index_m = rslt_base_index - rslt_row_offs; // i * DTT_SIZE2M1;
	rslt[rslt_base_index_p] = corr_pixscale * qdata0[i_transform_size]; // incomplete, will only be used for thread i=0
	rslt[rslt_base_index_m] = rslt[rslt_base_index_p];                  // nop for i=0 incomplete, will only be used for thread i=0
///	for (int j = 1; j < DTT_SIZE; j++) {
	for (int j = 1; j <= corr_radius; j++) {
		int rslt_base_index_pp = rslt_base_index_p + j;
		int rslt_base_index_pm = rslt_base_index_p - j;
		rslt[rslt_base_index_pp] = corr_pixscale * (
				 qdata0[i_transform_size + j] +
				 qdata1[i_transform_size + j -1]); // incomplete, will only be used for thread i=0
		rslt[rslt_base_index_pm] = corr_pixscale * (
				 qdata0[i_transform_size + j] +
				-qdata1[i_transform_size + j -1]); // incomplete, will only be used for thread i=0
	}
	if (i == 0) {
		return;
	}
///	im1_transform_size = i_transform_size - DTT_SIZE1; // already is calculated
	float d = corr_pixscale * qdata2[im1_transform_size];
	rslt[rslt_base_index_p] += d;
	rslt[rslt_base_index_m] -= d;
	for (int j = 1; j <= corr_radius; j++) {
		int rslt_base_index_pp = rslt_base_index_p + j;
		int rslt_base_index_pm = rslt_base_index_p - j;
		int rslt_base_index_mp = rslt_base_index_m + j;
		int rslt_base_index_mm = rslt_base_index_m - j;
		float d2 = corr_pixscale * qdata2[im1_transform_size + j];
		float d3 = corr_pixscale * qdata3[im1_transform_size + j -1];
		//rslt[rslt_base_index_mp], rslt[rslt_base_index_mp] are partially calculated in the cycle common with i=0
		rslt[rslt_base_index_mp] = rslt[rslt_base_index_pp] - d2 - d3;
		rslt[rslt_base_index_mm] = rslt[rslt_base_index_pm] - d2 + d3;
		rslt[rslt_base_index_pp] += d2 + d3;
		rslt[rslt_base_index_pm] += d2 - d3;
	}
}

__device__ void dttii_2d(
		float * clt_corr) // shared memory, [4][DTT_SIZE1][DTT_SIZE]
{
    // change to 16-32 threads?? in next iteration
    // vert pass (hor pass in Java, before transpose. Here transposed, no transform needed)
    for (int q = 0; q < 4; q++){
    	int is_sin = (q >> 1) & 1;
    	dttii_shared_mem_nonortho(clt_corr + q * (DTT_SIZE1 * DTT_SIZE) + threadIdx.x , DTT_SIZE1, is_sin); // vertical pass, thread is column
    }
    __syncthreads();

    // hor pass, corresponding to vert pass in Java
    for (int q = 0; q < 4; q++){
    	int is_sin = q & 1;
    	dttii_shared_mem_nonortho(clt_corr + (q * DTT_SIZE + threadIdx.x) * DTT_SIZE1 ,  1, is_sin); // horizontal pass, tread is row
    }
    __syncthreads();

}

__device__ void dttiv_color_2d(
		float * clt_tile,
		int color)
{
    dctiv_nodiverg( // all colors
			clt_tile + (DTT_SIZE1 * threadIdx.x), // [0][threadIdx.x], // pointer to start of row
			1); //int inc);
//	__syncthreads();// worsened
    if (color == BAYER_GREEN){
        dstiv_nodiverg( // all colors
				clt_tile + DTT_SIZE1 * threadIdx.x + DTT_SIZE1 * DTT_SIZE, // clt_tile[1][threadIdx.x], // pointer to start of row
    			1); //int inc);

    }
  	 __syncthreads();// __syncwarp();

#ifdef DEBUG222
    if ((threadIdx.x) == 0){
        printf("\nDTT Tiles after horizontal pass, color=%d\n",color);
    	debug_print_clt1(clt_tile, color, (color== BAYER_GREEN)?3:1); // only 1 quadrant for R,B and 2 - for G
    }
     __syncthreads();// __syncwarp();
#endif
    dctiv_nodiverg( // all colors
    		clt_tile + threadIdx.x, //  &clt_tile[0][0][threadIdx.x], // pointer to start of column
			DTT_SIZE1); // int inc,
//	__syncthreads();// worsened
    if (color == BAYER_GREEN){
          dctiv_nodiverg( // all colors
        		clt_tile + threadIdx.x + (DTT_SIZE1 * DTT_SIZE), // &clt_tile[1][0][threadIdx.x], // pointer to start of column
    			DTT_SIZE1); // int inc,
    }
  	 __syncthreads();// __syncwarp();
}

__device__ void dttiv_mono_2d(
		float * clt_tile)
{
	// Copy 0-> 1

    dctiv_nodiverg(
			clt_tile + (DTT_SIZE1 * threadIdx.x) + (0 * DTT_SIZE1 * DTT_SIZE),
			1); //int inc);
    dstiv_nodiverg(
    		clt_tile + (DTT_SIZE1 * threadIdx.x) + (1 * DTT_SIZE1 * DTT_SIZE),
			1); //int inc);
    dctiv_nodiverg(
			clt_tile + (DTT_SIZE1 * threadIdx.x) + (2 * DTT_SIZE1 * DTT_SIZE),
			1); //int inc);
    dstiv_nodiverg(
    		clt_tile + (DTT_SIZE1 * threadIdx.x) + (3 * DTT_SIZE1 * DTT_SIZE),
			1); //int inc);
	__syncthreads();// __syncwarp();

#ifdef DEBUG222
    if ((threadIdx.x) == 0){
        printf("\nDTT Tiles after horizontal pass, color=%d\n",color);
    	debug_print_clt1(clt_tile, color, (color== BAYER_GREEN)?3:1); // only 1 quadrant for R,B and 2 - for G
    }
     __syncthreads();// __syncwarp();
#endif

 	dctiv_nodiverg( // CC
 			clt_tile + threadIdx.x,
			DTT_SIZE1); // int inc,
 	dctiv_nodiverg( // SC
 			clt_tile + threadIdx.x + 1 * (DTT_SIZE1 * DTT_SIZE),
			DTT_SIZE1); // int inc,
 	dstiv_nodiverg( // CS
 			clt_tile + threadIdx.x + 2 * (DTT_SIZE1 * DTT_SIZE), // &clt_tile[1][0][threadIdx.x], // pointer to start of column
			DTT_SIZE1); // int inc,
 	dstiv_nodiverg( // SS
 			clt_tile + threadIdx.x + 3 * (DTT_SIZE1 * DTT_SIZE), // &clt_tile[1][0][threadIdx.x], // pointer to start of column
			DTT_SIZE1); // int inc,
  	 __syncthreads();// __syncwarp();
}



//
// Uses 16 threads, gets 4*8*8 clt tiles, performs idtt-iv (swapping 1 and 2 quadrants) and then unfolds with window,
// adding to the output 16x16 tile (to use Read-modify-write with 4 passes over the frame. Should be zeroed before the
// first pass
//__constant__ int imclt_indx9[16] = {0x28,0x31,0x3a,0x43,0x43,0x3a,0x31,0x28,0x1f,0x16,0x0d,0x04,0x04,0x0d,0x16,0x1f};
__device__ void imclt(
		float * clt_tile,   //        [4][DTT_SIZE][DTT_SIZE1], // +1 to alternate column ports [4][8][9]
		float * mclt_tile ) //           [2* DTT_SIZE][DTT_SIZE1+ DTT_SIZE], // +1 to alternate column ports[16][17]
{
	int thr3 =    threadIdx.x >> 3;
	int column =  threadIdx.x; // modify to use 2*8 threads, if needed.
	int thr012 =  threadIdx.x & 7;
	int column4 = threadIdx.x >> 2;
//	int wcolumn =column ^ (7 * thr3); //0..7,7,..0
//	int wcolumn = ((thr3 << 3) -1) ^ thr3; //0..7,7,..0
	int wcolumn = ((thr3 << 3) - thr3) ^ thr012; //0..7,7,..0
	float * clt_tile1 = clt_tile +  (DTT_SIZE1 * DTT_SIZE);
	float * clt_tile2 = clt_tile1 + (DTT_SIZE1 * DTT_SIZE);
	float * clt_tile3 = clt_tile2 + (DTT_SIZE1 * DTT_SIZE);
#ifdef DEBUG3
    if ((threadIdx.x) == 0){
        printf("\nDTT Tiles before IDTT\n");
    	debug_print_clt1(clt_tile, -1,  0xf); // only 1 quadrant for R,B and 2 - for G
    }
     __syncthreads();// __syncwarp();
#endif

	// perform horizontal dct-iv on quadrants 0 and 1
    dctiv_nodiverg(
    		clt_tile +  DTT_SIZE1 * (thr012 + 2*DTT_SIZE * thr3), // pointer to start of row for quadrants 0 and 2
			1);
	// perform horizontal dst-iv on quadrants 2 and 3
    dstiv_nodiverg( // all colors
    		clt_tile1 + DTT_SIZE1 * (thr012 + 2*DTT_SIZE * thr3), // pointer to start of row for quadrants 1 and 3
			1);
    __syncthreads();// __syncwarp();
	// perform vertical   dct-iv on quadrants 0 and 2
    dctiv_nodiverg(
    		clt_tile +  thr012 + (DTT_SIZE1 *   DTT_SIZE) * thr3, // pointer to start of row for quadrants 0 and 1
			DTT_SIZE1);
	// perform vertical   dst-iv on quadrants 1 and 3
    dstiv_nodiverg(
    		clt_tile2 + thr012 + (DTT_SIZE1 *   DTT_SIZE) * thr3, // pointer to start of row for quadrants 2 and 3
			DTT_SIZE1);
    __syncthreads();// __syncwarp();

#ifdef DEBUG3
    if ((threadIdx.x) == 0){
        printf("\nDTT Tiles after IDTT\n");
    	debug_print_clt1(clt_tile, -1,  0xf); // only 1 quadrant for R,B and 2 - for G
    }
     __syncthreads();// __syncwarp();
#endif


    float hw = HWINDOW2[wcolumn];
    int clt_offset = imclt_indx9[column]; // index in each of the 4 iclt quadrants, accounting for stride=9
    float * rslt = mclt_tile + column;
#pragma unroll
    for (int i = 0; i < 4; i++){
    	float val = *rslt;
    	float w = HWINDOW2[i] * hw;
    	float d0 = idct_signs[0][0][column4] * (*(clt_tile +  clt_offset));
    	float d1 = idct_signs[1][0][column4] * (*(clt_tile1 + clt_offset));
    	float d2 = idct_signs[2][0][column4] * (*(clt_tile2 + clt_offset));
    	float d3 = idct_signs[3][0][column4] * (*(clt_tile3 + clt_offset));
    	d0+=d1;
    	d2+=d3;
    	d0+= d2;
    	if (i < 3){
    		clt_offset +=  DTT_SIZE1;
    	}
//    	*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    	val = __fmaf_rd(w,d0,val); // w*d0 + val
    	*rslt = val;
    	rslt += DTT_SIZE21;
    }
#pragma unroll
    for (int i = 4; i < 8; i++){
    	float val = *rslt;
    	float w = HWINDOW2[i] * hw;
    	float d0 = idct_signs[0][1][column4] * (*(clt_tile +  clt_offset));
    	float d1 = idct_signs[1][1][column4] * (*(clt_tile1 + clt_offset));
    	float d2 = idct_signs[2][1][column4] * (*(clt_tile2 + clt_offset));
    	float d3 = idct_signs[3][1][column4] * (*(clt_tile3 + clt_offset));
    	d0+=d1;
    	d2+=d3;
    	d0+= d2;
//    	if (i < 7){
   		clt_offset -=  DTT_SIZE1;
//    	}
    	*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    	rslt += DTT_SIZE21;
    }
#pragma unroll
    for (int i = 7; i >= 4; i--){
    	float val = *rslt;
    	float w = HWINDOW2[i] * hw;
    	float d0 = idct_signs[0][2][column4] * (*(clt_tile +  clt_offset));
    	float d1 = idct_signs[1][2][column4] * (*(clt_tile1 + clt_offset));
    	float d2 = idct_signs[2][2][column4] * (*(clt_tile2 + clt_offset));
    	float d3 = idct_signs[3][2][column4] * (*(clt_tile3 + clt_offset));
    	d0+=d1;
    	d2+=d3;
    	d0+= d2;
    	if (i > 4){
    		clt_offset -=  DTT_SIZE1;
    	}
    	*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    	rslt += DTT_SIZE21;
    }
#pragma unroll
    for (int i = 3; i >= 0; i--){
    	float val = *rslt;
    	float w = HWINDOW2[i] * hw;
    	float d0 = idct_signs[0][3][column4] * (*(clt_tile +  clt_offset));
    	float d1 = idct_signs[1][3][column4] * (*(clt_tile1 + clt_offset));
    	float d2 = idct_signs[2][3][column4] * (*(clt_tile2 + clt_offset));
    	float d3 = idct_signs[3][3][column4] * (*(clt_tile3 + clt_offset));
    	d0+=d1;
    	d2+=d3;
    	d0+= d2;
    	if (i > 0){
    		clt_offset +=  DTT_SIZE1;
    	}
    	*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    	rslt += DTT_SIZE21;
    }
#ifdef DEBUG3
    __syncthreads();// __syncwarp();
    if ((threadIdx.x) == 0){
        printf("\nMCLT Tiles after IMCLT\n");
    	debug_print_mclt(mclt_tile, -1); // only 1 quadrant for R,B and 2 - for G
    }
    __syncthreads();// __syncwarp();
#endif
}


// Uses 8 threads, gets 4*8*8 clt tiles, performs idtt-iv (swapping 1 and 2 quadrants) and then unfolds to the 16x16
// adding to the output 16x16 tile (to use Read-modify-write with 4 passes over the frame. Should be zeroed before the
// first pass
//__constant__ int imclt_indx9[16] = {0x28,0x31,0x3a,0x43,0x43,0x3a,0x31,0x28,0x1f,0x16,0x0d,0x04,0x04,0x0d,0x16,0x1f};

__device__ void imclt8threads(
		int     do_acc,     // 1 - add to previous value, 0 - overwrite
		float * clt_tile,   //        [4][DTT_SIZE][DTT_SIZE1], // +1 to alternate column ports [4][8][9]
		float * mclt_tile,  //           [2* DTT_SIZE][DTT_SIZE1+ DTT_SIZE], // +1 to alternate column ports[16][17]
		int debug)
{
//	int thr3 =    threadIdx.x >> 3;
//	int column =  threadIdx.x; // modify to use 2*8 threads, if needed.
//	int thr012 =  threadIdx.x & 7;
//	int column4 = threadIdx.x >> 2;
//	int wcolumn = ((thr3 << 3) - thr3) ^ thr012; //0..7,7,..0
	float * clt_tile1 = clt_tile +  (DTT_SIZE1 * DTT_SIZE);
	float * clt_tile2 = clt_tile1 + (DTT_SIZE1 * DTT_SIZE);
	float * clt_tile3 = clt_tile2 + (DTT_SIZE1 * DTT_SIZE);
#ifdef DEBUG7
    if (debug && (threadIdx.x == 0) && (threadIdx.y == 0)){
        printf("\nDTT Tiles before IDTT\n");
        debug_print_clt_scaled(clt_tile, -1,  0xf, 0.25); // only 1 quadrant for R,B and 2 - for G
    }
     __syncthreads();// __syncwarp();
#endif

	// perform horizontal dct-iv on quadrants 0 and 1
    dctiv_nodiverg( // quadrant 0
    		clt_tile +  threadIdx.x,                              // pointer to start of row for quadrant 0
			DTT_SIZE1);
    dctiv_nodiverg( // quadrant 1
    		clt_tile +  threadIdx.x + (1 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 1
			DTT_SIZE1);
	// perform horizontal dst-iv on quadrants 2 and 3
    dstiv_nodiverg( // quadrant 2
    		clt_tile +  threadIdx.x + (2 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 2
			DTT_SIZE1);
    dstiv_nodiverg( // quadrant 3
    		clt_tile +  threadIdx.x + (3 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 3
			DTT_SIZE1);
    __syncthreads();// __syncwarp();
	// perform vertical   dct-iv on quadrants 0 and 2
    dctiv_nodiverg( // quadrant 0
    		clt_tile +  DTT_SIZE1 * threadIdx.x,                              // pointer to start of row for quadrant 0
			1);
    dctiv_nodiverg( // quadrant 2
    		clt_tile +  DTT_SIZE1 * threadIdx.x + (2 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 2
			1);
    // perform vertical   dst-iv on quadrants 1 and 3
    dstiv_nodiverg( // quadrant 1
    		clt_tile +  DTT_SIZE1 * threadIdx.x + (1 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 1
			1);
    dstiv_nodiverg( // quadrant 3
    		clt_tile +  DTT_SIZE1 * threadIdx.x + (3 * DTT_SIZE * DTT_SIZE1), // pointer to start of row for quadrant 3
			1);
    __syncthreads();// __syncwarp();

#ifdef DEBUG7
    if (debug && (threadIdx.x == 0) && (threadIdx.y == 0)){
    	printf("\nDTT Tiles after IDTT\n");
    	debug_print_clt_scaled(clt_tile, -1,  0xf, 0.25); // only 1 quadrant for R,B and 2 - for G
    }
    __syncthreads();// __syncwarp();
#endif
    // re-using 16-thread code (thr3 was bit 3 of threadIdx.x).
    for (int thr3 = 0; thr3 < 2; thr3++){
    	int thr3m = (thr3 << 3);
    	int column =  threadIdx.x + thr3m; // modify to use 2*8 threads, if needed.
    	int thr012 =  threadIdx.x & 7; // == threadIdx.x
    	int column4 = column >> 2; // (threadIdx.x >> 2) | (thr3 << 1) ; // different !
    	int wcolumn = (thr3m - thr3) ^ thr012; //0..7,7,..0

    	float hw = HWINDOW2[wcolumn];
    	int clt_offset = imclt_indx9[column]; // index in each of the 4 iclt quadrants, accounting for stride=9
    	float * rslt = mclt_tile + column;
#ifdef DEBUG7
        if (debug && (threadIdx.x == 0) && (threadIdx.y == 0)){
    	printf("\nUnrolling: thr3=%d, thr3m=%d, column=%d, thr012=%d, column4=%d, wcolumn=%d, hw=%f, clt_offset=%d\n",
    			thr3, thr3m, column, thr012, column4, wcolumn, hw, clt_offset);
    	debug_print_clt1(clt_tile, -1,  0xf); // only 1 quadrant for R,B and 2 - for G
    }
    __syncthreads();// __syncwarp();
#endif

#pragma unroll
    	for (int i = 0; i < 4; i++){
    		float val = *rslt;
    		// facc
    		float w = HWINDOW2[i] * hw;
    		float d0 = idct_signs[0][0][column4] * (*(clt_tile +  clt_offset));
    		float d1 = idct_signs[1][0][column4] * (*(clt_tile1 + clt_offset));
    		float d2 = idct_signs[2][0][column4] * (*(clt_tile2 + clt_offset));
    		float d3 = idct_signs[3][0][column4] * (*(clt_tile3 + clt_offset));
    		d0+=d1;
    		d2+=d3;
    		d0+= d2;
    		if (i < 3){
    			clt_offset +=  DTT_SIZE1;
    		}
    		//    	*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    		// val =__fmaf_rd(w,d0,val); // w*d0 + val
    		// *rslt = val;
    		*rslt = do_acc? __fmaf_rd(w,d0,val) : w * d0; // w*d0 + val do_acc - common for all thereads
    		rslt += DTT_SIZE21;
    	}
#pragma unroll
    	for (int i = 4; i < 8; i++){
    		float val = *rslt;
    		float w = HWINDOW2[i] * hw;
    		float d0 = idct_signs[0][1][column4] * (*(clt_tile +  clt_offset));
    		float d1 = idct_signs[1][1][column4] * (*(clt_tile1 + clt_offset));
    		float d2 = idct_signs[2][1][column4] * (*(clt_tile2 + clt_offset));
    		float d3 = idct_signs[3][1][column4] * (*(clt_tile3 + clt_offset));
    		d0+=d1;
    		d2+=d3;
    		d0+= d2;
    		//    	if (i < 7){
    		clt_offset -=  DTT_SIZE1;
    		//    	}
//    		*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    		*rslt = do_acc? __fmaf_rd(w,d0,val) : w * d0; // w*d0 + val do_acc - common for all thereads

    		rslt += DTT_SIZE21;
    	}
#pragma unroll
    	for (int i = 7; i >= 4; i--){
    		float val = *rslt;
    		float w = HWINDOW2[i] * hw;
    		float d0 = idct_signs[0][2][column4] * (*(clt_tile +  clt_offset));
    		float d1 = idct_signs[1][2][column4] * (*(clt_tile1 + clt_offset));
    		float d2 = idct_signs[2][2][column4] * (*(clt_tile2 + clt_offset));
    		float d3 = idct_signs[3][2][column4] * (*(clt_tile3 + clt_offset));
    		d0+=d1;
    		d2+=d3;
    		d0+= d2;
    		if (i > 4){
    			clt_offset -=  DTT_SIZE1;
    		}
    		//*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    		*rslt = do_acc? __fmaf_rd(w,d0,val) : w * d0; // w*d0 + val do_acc - common for all thereads
    		rslt += DTT_SIZE21;
    	}
#pragma unroll
    	for (int i = 3; i >= 0; i--){
    		float val = *rslt;
    		float w = HWINDOW2[i] * hw;
    		float d0 = idct_signs[0][3][column4] * (*(clt_tile +  clt_offset));
    		float d1 = idct_signs[1][3][column4] * (*(clt_tile1 + clt_offset));
    		float d2 = idct_signs[2][3][column4] * (*(clt_tile2 + clt_offset));
    		float d3 = idct_signs[3][3][column4] * (*(clt_tile3 + clt_offset));
    		d0+=d1;
    		d2+=d3;
    		d0+= d2;
    		if (i > 0){
    			clt_offset +=  DTT_SIZE1;
    		}
    		//*rslt = __fmaf_rd(w,d0,val); // w*d0 + val
    		*rslt = do_acc? __fmaf_rd(w,d0,val) : w * d0; // w*d0 + val do_acc - common for all thereads
    		rslt += DTT_SIZE21;
    	}
    }
#ifdef DEBUG7
    __syncthreads();// __syncwarp();
	for (int ccam = 0; ccam < NUM_CAMS; ccam++) {
		if (debug  && (threadIdx.x == 0) && (threadIdx.y == ccam)){
			printf("\nMCLT Tiles after IMCLT, cam=%d\n", threadIdx.y);
			debug_print_mclt(
					mclt_tile, //         [4][DTT_SIZE][DTT_SIZE1], // +1 to alternate column ports)
					-1);
		}
		__syncthreads();// __syncwarp();
	}
    __syncthreads();// __syncwarp();
#endif
}




//#endif

